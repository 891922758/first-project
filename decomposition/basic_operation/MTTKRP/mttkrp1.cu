#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include "GPUTimer.h"
#include <hipblas.h>
#include <hipsolver.h>
#include <hiprand.h>

using namespace std;
typedef float dt;

__global__  void floattohalf(dt *AA,half *BB,long m){
	long i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<m){
		BB[i]=__float2half(AA[i]);
	}
	__syncthreads();
}

void f2h(dt *A,half *B,long num){
	dim3 threads(512,1,1);
	dim3 blocks((num+512-1)/512,1,1);	
	floattohalf<<<blocks,threads>>>(A,B,num);
}

__global__ void krpro(dt *M,dt *N,dt *res,long long m,long long n,long long r){
	//m*r and n*r to (m*n)*r	
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;
	while(i<m*n*r){
		long long row = i%(m*n);
		long long col = i/(m*n);
		res[col*m*n+row] = M[(row/n)+col*m]*N[(row%n)+col*n];
		i+=temp;
	}
    __syncthreads();
}
__global__ void tensorToMode1(dt *T1,dt *T2,int m,int n,int k ){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<m*n*k){
		int tube = i/(m*n);
		int row = (i-tube*(m*n))%m;
		int col = (i-tube*(m*n))/m;
		T2[tube*m*n+col*m+row] = T1[tube*m*n+col*m+row];
	}
	__syncthreads();
	
}

void printTensor(dt *A,int a,int b,int c){
	dt *h_A;
	hipHostAlloc((void**)&h_A,sizeof(dt)*a*b*c,0);
	hipMemcpyAsync(h_A,A,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost,0);
	hipDeviceSynchronize();

	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<h_A[i*a*b+k*a+j]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
	hipHostFree(h_A);
}

int main(int argc,char *argv[]){
for(int i = 160;i<1400;i=i+160){
	int m = i;
	int n = m;
	int k = m;
	int r = (int)(m*0.1);
//	int r = m;
	cout<<m<<endl;
	size_t size=sizeof(dt);
	float time1 = 0.0;
	float time2 = 0.0;
	float time3 = 0.0;
	float temp = 0.0;
	dt *X,*B,*C;
	hipHostAlloc((void**)&X,size*m*n*k,0);
	hipHostAlloc((void**)&B,size*n*r,0);
	hipHostAlloc((void**)&C,size*k*r,0);
//	srand(2);
	for(int i = 0;i<m*n*k;i++){
		X[i] = (((float) rand())/RAND_MAX - 0.5);
	}
	for(int i = 0;i<n*r;i++){
		B[i] = (((float) rand())/RAND_MAX - 0.5);
	}
	for(int i = 0;i<k*r;i++){
		C[i] = (((float) rand())/RAND_MAX - 0.5);
	}
	
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	GPUTimer timer;
	
	dt *d_X;
	hipMalloc((void**)&d_X,size*m*n*k);
	hipMemcpyAsync(d_X,X,size*m*n*k,hipMemcpyHostToDevice,0);
	dt *d_CkrB,*d_C,*d_B,*d_result;
	hipMalloc((void**)&d_B,size*n*r);
	hipMalloc((void**)&d_C,size*k*r);
	hipMalloc((void**)&d_result,size*m*r);
	hipMemcpyAsync(d_C,C,size*k*r,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_B,B,size*n*r,hipMemcpyHostToDevice,0);
	hipDeviceSynchronize();
	hipMalloc((void**)&d_CkrB,size*n*k*r);
	half *h_CkrB,*h_X;
	hipMalloc((void **)&h_X,sizeof(half)*m*n*k);
	hipMalloc((void **)&h_CkrB,sizeof(half)*n*k*r);

	dim3 thread(512,1,1);
	dim3 block((m*n*k+512-1)/512,1,1); //for tensor matrix
	dim3 block1((n*k*r+512-1)/512,1,1); //for kr

//	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	hipblasSetMathMode(handle,HIPBLAS_DEFAULT_MATH);
	int L = 10;

for(int iter = 0;iter<L;++iter){
	//warm up
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,n,k,1,&alpha,d_B,n,n,d_C,k,k,&beta,d_CkrB,n,k*n,r);
	hipDeviceSynchronize();
//	cout<<"d_CkrB"<<endl;
//	printTensor(d_CkrB,k*n,r,1);

	dt *d_X1;
	hipMalloc((void**)&d_X1,size*m*n*k);

	timer.start();
	krpro<<<block1,thread>>>(d_C,d_B,d_CkrB,k,n,r);
	hipDeviceSynchronize();
//	cout<<"unopt-d_CkrB"<<endl;
//	printTensor(d_CkrB,k*n,r,1);
	tensorToMode1<<<block,thread>>>(d_X,d_X1,m,n,k);
	hipDeviceSynchronize();
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,r,n*k,&alpha,d_X1,m,d_CkrB,n*k,&beta,d_result,m);
	hipDeviceSynchronize();
	time1 = time1+timer.seconds();
//	cout<<"unopt-d_result"<<endl;
//	printTensor(d_result+2*m,3,4,1);
//	printTensor(d_result,m,r,1);
	hipFree(d_X1);
	
	timer.start();
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,n,k,1,&alpha,d_B,n,n,d_C,k,k,&beta,d_CkrB,n,k*n,r);
	hipDeviceSynchronize();
//	cout<<"opt-d_CkrB"<<endl;
//	printTensor(d_CkrB,k*n,r,1);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,r,n*k,&alpha,d_X,m,d_CkrB,n*k,&beta,d_result,m);
	hipDeviceSynchronize();
	time2 =time2+timer.seconds();
//	cout<<"opt-d_result"<<endl;
//	printTensor(d_result+2*m,3,4,1);
//	printTensor(d_result,m,r,1);

	timer.start();
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,n,k,1,&alpha,d_B,n,n,d_C,k,k,&beta,d_CkrB,n,k*n,r);
	hipDeviceSynchronize();
	temp = timer.seconds();
//	cout<<"opt-d_CkrB"<<endl;
//	printTensor(d_CkrB,k*n,r,1);
	f2h(d_CkrB,h_CkrB,k*n*r);
	f2h(d_X,h_X,m*n*k);
	hipDeviceSynchronize();
	timer.start();
	hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,r,n*k,&alpha,h_X,HIP_R_16F,m,h_CkrB,HIP_R_16F,n*k,&beta,d_result,HIP_R_32F,m,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,r,n*k,&alpha,h_X,HIP_R_16F,m,h_CkrB,HIP_R_16F,n*k,&beta,d_result,HIP_R_32F,m,HIP_R_32F,HIPBLAS_GEMM_DEFAULT);
	hipDeviceSynchronize();
	time3 =time3+ temp+timer.seconds();
//	cout<<"opt-d_result"<<endl;
//	printTensor(d_result+2*m,3,4,1);
//	printTensor(d_result,m,r,1);

	if(iter == L-1){
		cout<<"mode1-unop = "<<time1/L<<"ms"<<endl;
		cout<<"mode1-op = "<<time2/L<<"ms"<<endl;
		cout<<"mode1-op-tensor-core = "<<time3/L<<"ms"<<endl;
	}
}

	hipblasDestroy(handle);	

	hipHostFree(X);
	hipHostFree(B);
	hipHostFree(C);
	hipFree(h_X);
	hipFree(h_CkrB);
	hipFree(d_X);
	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_CkrB);
	hipFree(d_result);
}
	return 0;
}
