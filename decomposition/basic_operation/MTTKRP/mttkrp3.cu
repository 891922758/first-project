#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "GPUTimer.h"
#include <hipblas.h>
#include <hipsolver.h>
#include <hiprand.h>

using namespace std;
typedef float dt;

__global__  void floattohalf(dt *AA,half *BB,long m){
	long i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<m){
		BB[i]=__float2half(AA[i]);
	}
	__syncthreads();
}

void f2h(dt *A,half *B,long num){
	dim3 threads(512,1,1);
	dim3 blocks((num+512-1)/512,1,1);	
	floattohalf<<<blocks,threads>>>(A,B,num);
}

__global__ void krpro(dt *M,dt *N,dt *res,long long m,long long n,long long r){
	//m*r and n*r to (m*n)*r	
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	const long long temp = blockDim.x*gridDim.x;
	while(i<m*n*r){
		long long row = i%(m*n);
		long long col = i/(m*n);
		res[col*m*n+row] = M[(row/n)+col*m]*N[(row%n)+col*n];
		i+=temp;
	}
    __syncthreads();
}

__global__ void tensorToMode3(dt *T1,dt *T2,int m,int n,int k){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<m*n*k){
		int tube = i/(m*n);
		int row = (i-tube*(m*n))%m;
		int col = (i-tube*(m*n))/m;
		T2[k*(col*m+row)+tube] = T1[tube*m*n+col*m+row];
	}
    __syncthreads();
}


void printTensor(dt *A,int a,int b,int c){
	dt *h_A;
	hipHostAlloc((void**)&h_A,sizeof(dt)*a*b*c,0);
	hipMemcpyAsync(h_A,A,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost,0);
	hipDeviceSynchronize();

	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<h_A[i*a*b+k*a+j]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
	hipHostFree(h_A);
}

int main(int argc,char *argv[]){
for(int i = 160;i<1600;i=i+160){
	int a = i;
	int b = a;
	int c = a;
	int r = (int)(a*0.1);
//	int r = 2;
	cout<<a<<endl;
	size_t size=sizeof(dt);
	float time1 = 0.0;
	float time2 = 0.0;
	float time3 = 0.0;
	float temp = 0.0;
	dt *A,*B,*X;
	hipHostAlloc((void**)&X,size*a*b*c,0);
	hipHostAlloc((void**)&B,size*b*r,0);
	hipHostAlloc((void**)&A,size*a*r,0);

	for(int i = 0;i<a*b*c;i++){
		X[i] = (((float) rand())/RAND_MAX - 0.5);
	}
	for(int i = 0;i<b*r;i++){
		B[i] = (((float) rand())/RAND_MAX - 0.5);
	}
	for(int i = 0;i<a*r;i++){
		A[i] = (((float) rand())/RAND_MAX - 0.5);
	}
	
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	GPUTimer timer;
	
	dt *d_X;
	hipMalloc((void**)&d_X,size*a*b*c);
	hipMemcpyAsync(d_X,X,size*a*b*c,hipMemcpyHostToDevice,0);
	dt *d_BkrA,*d_A,*d_B,*d_result;
	hipMalloc((void**)&d_B,size*b*r);
	hipMalloc((void**)&d_A,size*a*r);
	hipMalloc((void**)&d_result,size*c*r);
	hipMemcpyAsync(d_A,A,size*a*r,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_B,B,size*b*r,hipMemcpyHostToDevice,0);
	hipDeviceSynchronize();
	hipMalloc((void**)&d_BkrA,size*a*b*r);
	half *h_BkrA,*h_X;
	hipMalloc((void **)&h_X,sizeof(half)*a*b*c);
	hipMalloc((void **)&h_BkrA,sizeof(half)*a*b*r);

	dim3 thread(512,1,1);
	dim3 block((a*b*c+512-1)/512,1,1);// for tensor matrix
	dim3 block1((a*b*r+512-1)/512,1,1);//for kr

//	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	hipblasSetMathMode(handle,HIPBLAS_DEFAULT_MATH);
	int L = 10;

for(int iter = 0;iter<L;++iter){
	//warm up
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,b*a,r);
	hipDeviceSynchronize();
	
	dt *d_X3;
	hipMalloc((void**)&d_X3,size*a*b*c);

	timer.start();
	krpro<<<block1,thread>>>(d_B,d_A,d_BkrA,b,a,r);
	hipDeviceSynchronize();
//	cout<<"unopt-d_BkrA"<<endl;
//	printTensor(d_BkrA,a*b,r,1);
	tensorToMode3<<<block,thread>>>(d_X,d_X3,a,b,c);
	hipDeviceSynchronize();
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,c,r,a*b,&alpha,d_X3,c,d_BkrA,a*b,&beta,d_result,c);
	hipDeviceSynchronize();
	time1 = time1+timer.seconds();
//	cout<<"unopt-d_result"<<endl;
//	printTensor(d_result+2*c,2,3,1);
//	printTensor(d_result,c,r,1);
	hipFree(d_X3);

	timer.start();
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,b*a,r);
	hipDeviceSynchronize();
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,r,a*b,&alpha,d_X,a*b,d_BkrA,a*b,&beta,d_result,c);
	hipDeviceSynchronize();
	time2 = time2+timer.seconds();
//	cout<<"opt-d_result"<<endl;
//	printTensor(d_result+2*c,2,3,1);
//	printTensor(d_result,c,r,1);

	timer.start();
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,b*a,r);
	hipDeviceSynchronize();
	temp = timer.seconds();
	f2h(d_BkrA,h_BkrA,a*b*r);
	f2h(d_X,h_X,a*b*c);
	hipDeviceSynchronize();
	timer.start();
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,r,a*b,&alpha,h_X,HIP_R_16F,a*b,h_BkrA,HIP_R_16F,a*b,&beta,d_result,HIP_R_32F,c,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	hipDeviceSynchronize();
	time3 = time3+temp+timer.seconds();
//	cout<<"opt-d_result"<<endl;
//	printTensor(d_result+2*c,2,3,1);
//	printTensor(d_result,c,r,1);

	if(iter == L-1){
		cout<<"mode3-unop = "<<time1/L<<"ms"<<endl;
		cout<<"mode3-op = "<<time2/L<<"ms"<<endl;
		cout<<"mode3-op-tensorcore = "<<time3/L<<"ms"<<endl;
	}
}

	hipblasDestroy(handle);	

	hipHostFree(X);
	hipHostFree(B);
	hipHostFree(A);
	hipFree(h_X);
	hipFree(h_BkrA);
	hipFree(d_X);
	hipFree(d_B);
	hipFree(d_A);
	hipFree(d_BkrA);
	hipFree(d_result);
}
	return 0;
}
