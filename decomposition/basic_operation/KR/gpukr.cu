#include <iostream>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <time.h>
 
typedef float dt;
using namespace std;

__global__ void krpro(dt *M,dt *N,dt *res,long long m,long long n,long long r){
	//m*r and n*r to (m*n)*r	
	long long i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<m*n*r){
		long long row = i%(m*n);
		long long col = i/(m*n);
		res[col*m*n+row] = M[(row/n)+col*m]*N[(row%n)+col*n];
	}
    __syncthreads();
}

void printTensor(dt *A,int a,int b,int c){
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<A[i*a*b+k*a+j]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
}

int main(int argc,char *argv[]){
for(int hh = 160;hh<=1600;hh=hh+160){
	long long a = hh;
	long long b = a;
	long long r = a;	//a*r b*r
	dt *A,*B;
	cout<<a<<endl;
	hipHostAlloc((void**)&A,sizeof(dt)*a*r,0);
	hipHostAlloc((void**)&B,sizeof(dt)*b*r,0);
	srand(5);
	for(long long i = 0;i<a*r;i++){
		A[i] = rand()*0.1/(RAND_MAX*0.1);		//initial Tensor A
	}
	for(long long i = 0;i<b*r;i++){
		B[i] = rand()*0.1/(RAND_MAX*0.1);		//initial Tensor A
	}
//printTensor(A,a,r,1);
//printTensor(B,b,r,1);
	dt *AkrB;
	hipHostAlloc((void**)&AkrB,sizeof(dt)*a*b*r,0);

	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dim3 threads(512,1,1);
	dim3 blocks((a*b*r+512-1)/512,1,1);
//	dim3 blocks(1024,1,1);
//	cout<<(a*b*r+1023)/1024<<endl;
	dt *d_A;
	dt *d_B;
	dt *d_AkrB;
	hipMalloc((void **)&d_A,sizeof(dt)*a*r);
	hipMalloc((void **)&d_B,sizeof(dt)*b*r);
	hipMalloc((void **)&d_AkrB,sizeof(dt)*a*b*r);
	hipMemcpyAsync(d_A,A,sizeof(dt)*a*r,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_B,B,sizeof(dt)*b*r,hipMemcpyHostToDevice,0);
	hipDeviceSynchronize();

//warm up
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,a,1,&alpha,d_B,b,b,d_A,a,a,&beta,d_AkrB,b,b*a,r);
	krpro<<<blocks,threads>>>(d_A,d_B,d_AkrB,a,b,r);

	hipEvent_t start,stop;
	dt elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
for(int j = 0;j<10;j++){
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,a,1,&alpha,d_B,b,b,d_A,a,a,&beta,d_AkrB,b,b*a,r);
//	krpro<<<blocks,threads>>>(d_A,d_B,d_AkrB,a,b,r);
}
	hipEventRecord(stop,0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	cout<<elapsedTime/10<<endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(d_A);
	hipFree(d_B);
	hipMemcpyAsync(AkrB,d_AkrB,sizeof(dt)*a*b*r,hipMemcpyDeviceToHost,0);
	hipDeviceSynchronize();
	hipFree(d_AkrB);
	hipblasDestroy(handle);	

//printTensor(AkrB,a*b,r,1);
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(AkrB);
}
	return 0;
}


