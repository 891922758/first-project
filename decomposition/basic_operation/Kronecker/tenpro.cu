
#include <iostream>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
 
typedef float dt;
using namespace std;

__global__ void kron(dt *M,dt *N,dt *res,long  m, long n,long k,long l){
	 long i = blockIdx.x*blockDim.x+threadIdx.x;
	if(i<m*n*k*l){
		long row = i%(m*k);
		long col = i/(m*k);
		res[col*m*k+row] = M[(row/k)+(col/l)*m]*N[(row%k)+(col%l)*k];
	}
    __syncthreads();
}


__global__ void obtainA(dt *M,dt *res,long  m, long n,long k,long l){
	 long i = blockIdx.x*blockDim.x+threadIdx.x;
	const  long temp = blockDim.x*gridDim.x;
	while(i<m*n*l){
		res[i] = M[m*(i/(m*l))+(i%m)];
		i+=temp;
	}
    __syncthreads();
}

__global__ void obtainB(dt *M,dt *res,long  m, long n,long k,long l){
	 long i = blockIdx.x*blockDim.x+threadIdx.x;
	const  long temp = blockDim.x*gridDim.x;
	while(i<n*k*l){
		res[i] = M[i%(k*l)];
		i+=temp;
	}
    __syncthreads();
}

void printTensor(dt *A,int a,int b,int c){
	for(int i = 0;i<c;i++){
		for(int j = 0;j<a;j++){
			for(int k =0;k<b;k++){
				cout<<A[i*a*b+k*a+j]<<"  ";
			}
			cout<<endl;
		}
		cout<<"-----------------------------------"<<endl;
	}
	cout<<endl;
}

int main(int argc,char *argv[]){
	hipSetDevice(0);

	for(int hh = 20;hh<=180;hh=hh+20){
		long a = hh;
		long b = a;
		long c = a;
		long d = a;
		dt *A,*B,*C;
		cout<<a<<" "<<b<<" "<<c<<" "<<d<<endl;
		hipHostAlloc((void**)&A,sizeof(dt)*a*b,0);
		hipHostAlloc((void**)&B,sizeof(dt)*c*d,0);
		hipHostAlloc((void**)&C,sizeof(dt)*a*b*c*d,0);
		for(long i = 0;i<a*b;i++){
			A[i] = rand()*0.1/(RAND_MAX*0.1);		//initial Tensor A
		}
		for(long i = 0;i<c*d;i++){
			B[i] = rand()*0.1/(RAND_MAX*0.1);		//initial Tensor A
		}
		for(long i = 0;i<a*b*c*d;i++){
			C[i] = rand()*0.1/(RAND_MAX*0.1);		//initial Tensor A
		}
//printTensor(A,a,b,1);
//printTensor(B,c,d,1);
		dt alpha = 1.0;
		dt beta = 0.0;
		hipblasHandle_t handle;
		hipblasCreate(&handle);
		dim3 threads(512,1,1);
		dim3 blocks((a*b*c*d+512-1)/512,1,1);
		dim3 bls(b,1,1);
		dt *d_A;
		dt *d_B;
		dt *d_C;
		hipMalloc((void **)&d_A,sizeof(dt)*a*b);
		hipMalloc((void **)&d_B,sizeof(dt)*c*d);
		hipMalloc((void **)&d_C,sizeof(dt)*a*b*c*d);
		hipMemcpyAsync(d_A,A,sizeof(dt)*a*b,hipMemcpyHostToDevice,0);
		hipMemcpyAsync(d_B,B,sizeof(dt)*c*d,hipMemcpyHostToDevice,0);
		hipDeviceSynchronize();
		//warm up
		kron<<<blocks,threads>>>(d_A,d_B,d_C,a,b,c,d);
		kron<<<blocks,threads>>>(d_A,d_B,d_C,a,b,c,d);
	hipEvent_t start,stop;
	dt elapsedTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	int L = 10;
	for(int j = 0;j<L;j++){
	//	for(int i = 0;i<b;i++){
	//		cublasSgemmStridedBatched(handle,CUBLAS_OP_N,CUBLAS_OP_T,c,a,1,&alpha,d_B,c,c,d_A+i*a,a,0,&beta,d_C+i*a*c*d,c,c*a,d);
	//}

		kron<<<blocks,threads>>>(d_A,d_B,d_C,a,b,c,d);
}
	hipEventRecord(stop,0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	cout<<elapsedTime/L<<endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);

		hipFree(d_A);
		hipFree(d_B);
		hipMemcpyAsync(C,d_C,sizeof(dt)*a*b*c*d,hipMemcpyDeviceToHost,0);
		hipDeviceSynchronize();
//printTensor(C+100,10,1,1);
		hipFree(d_C);

		hipblasDestroy(handle);	

		hipHostFree(A);
		hipHostFree(B);
		hipHostFree(C);
	
	}
	
	return 0;
}
