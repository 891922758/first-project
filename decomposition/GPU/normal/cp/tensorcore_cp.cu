#include "hip/hip_runtime.h"
#include "head.h"

void cp_tensorcore(dt *X,dt *A,dt *B,dt *C,long a,long b,long c,long r){
// X is a*b*c; A is a*r; B is b*r; C is c*r
// we assume they all store as column
	dt alpha = 1.0;
//	dt alpha1 = -1.0;
	dt beta = 0.0;
	dt sh=0.0;
	dt xia=1.0;
	dim3 threads(512,1,1);
	dim3 block1((r*r+512-1)/512,1,1); //for elepro
	dim3 block2((a*b*c+512-1)/512,1,1);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
//	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
//	int info = 0;
	int *d_info = NULL;
	hipMalloc((void**)&d_info,sizeof(int));
//	int *Ipiv; 
//	hipHostAlloc((void**)&Ipiv,sizeof(int)*r,0);
	int *d_Ipiv = NULL; // PA=LU, P is control weather permute
	hipMalloc((void**)&d_Ipiv,sizeof(int));
	int lwork=0;
	dt *d_work = NULL;

	dt *d_X,*d_C,*d_B,*d_A;
	hipMalloc((void**)&d_B,sizeof(dt)*b*r);
	hipMalloc((void**)&d_C,sizeof(dt)*c*r);
	hipMalloc((void**)&d_X,sizeof(dt)*a*b*c);
	hipMalloc((void**)&d_A,sizeof(dt)*a*r);

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	hiprandGenerateUniform(gen,d_B,b*r);
	hiprandGenerateUniform(gen,d_C,c*r);
	//hipMemcpyAsync(d_B,B,sizeof(dt)*b*r,hipMemcpyHostToDevice,0);
	//hipMemcpyAsync(d_C,C,sizeof(dt)*c*r,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_X,X,sizeof(dt)*a*b*c,hipMemcpyHostToDevice,0);
//	cout<<"B"<<endl; printTensor(d_B,b,r,1);
//	cout<<"C"<<endl; printTensor(d_C,c,r,1);
//	cout<<"X"<<endl; printTensor(d_X,a,b,c);

	dt *d_X2;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c);
	tensorToMode2<<<block2,threads>>>(d_X,d_X2,a,b,c);
	hipDeviceSynchronize();
//	cout<<"X2"<<endl; printTensor(d_X2,b,a,c);

	half *h_X;
	half *h_X2;
	hipMalloc((void**)&h_X,sizeof(half)*a*b*c);
	hipMalloc((void**)&h_X2,sizeof(half)*a*b*c);
	f2h(d_X,h_X,a*b*c);
	f2h(d_X2,h_X2,a*b*c);

	
	dt *d_ATA,*d_BTB,*d_CTC;
	hipMalloc((void**)&d_ATA,sizeof(dt)*r*r);
	hipMalloc((void**)&d_BTB,sizeof(dt)*r*r);
	hipMalloc((void**)&d_CTC,sizeof(dt)*r*r);

	dt *d_CkrB;
	hipMalloc((void**)&d_CkrB,sizeof(dt)*b*c*r); //GPU store C kr B
	dt *d_CkrA;
	hipMalloc((void**)&d_CkrA,sizeof(dt)*a*c*r); //GPU store C kr A
	dt *d_BkrA;
	hipMalloc((void**)&d_BkrA,sizeof(dt)*a*b*r); //GPU store B kr A

	dt *d_At_r;
	hipMalloc((void**)&d_At_r,sizeof(dt)*a*r); //GPU store (CkrB)'*X1' as right part 
	dt *d_At_l;
	hipMalloc((void**)&d_At_l,sizeof(dt)*r*r); //GPU store (CTC.*BTB)' as left part
	dt *d_Bt_r;
	hipMalloc((void**)&d_Bt_r,sizeof(dt)*b*r); //GPU store (CkrA)'*X2' as right part 
	dt *d_Bt_l;
	hipMalloc((void**)&d_Bt_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	dt *d_Ct_r;
	hipMalloc((void**)&d_Ct_r,sizeof(dt)*c*r); //GPU store (BkrA)'*X3' as right part 
	dt *d_Ct_l;
	hipMalloc((void**)&d_Ct_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	half *h_CkrB,*h_CkrA,*h_BkrA;
	hipMalloc((void**)&h_CkrB,sizeof(half)*c*b*r);
	hipMalloc((void**)&h_CkrA,sizeof(half)*a*c*r);
	hipMalloc((void**)&h_BkrA,sizeof(half)*a*b*r);

	const int L = 10;
for(int i = 0;i<L;i++){
// update A
//	cout<<"unpdta A"<<endl;
// we compute kr(dot) product
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,c,1,&alpha,d_B,b,b,d_C,c,c,&beta,d_CkrB,b,b*c,r);
//	cout<<"CkrB"<<endl; printTensor(d_CkrB,c*b,r,1);
	f2h(d_CkrB,h_CkrB,b*c*r);
// compute (CkrB)'*X1'
//	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,a,b*c,&alpha,d_CkrB,b*c,d_X,a,&beta,d_At_r,r);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,a,b*c,&alpha,h_CkrB,HIP_R_16F,b*c,h_X,HIP_R_16F,a,&beta,d_At_r,HIP_R_32F,r,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	cout<<"CkrB'*X1'"<<endl;printTensor(d_At_r,r,a,1);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,c,&alpha,d_C,c,d_C,c,&beta,d_CTC,r);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);
//	cout<<"CTC"<<endl; printTensor(d_CTC,r,r,1);
//	cout<<"BTB"<<endl; printTensor(d_BTB,r,r,1);
	// compute (CTC.*BTB)'  
	elepro<<<block1,threads>>>(d_CTC,d_BTB,d_At_l,r*r);
//	cout<<"CTC.*BTB"<<endl;printTensor(d_At_l,r,r,1);

	//then we solve least squares minimization
	// (d_At_l)'A'=d_At_r ,due to d_At_l is symc so we don't tran, d_At_r has implity tran
	// r*r     r*a    r*a store as col
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_At_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_At_l,r,d_work,d_Ipiv,d_info);
//	hipMemcpy(Ipiv,d_Ipiv,sizeof(int)*r,hipMemcpyDeviceToHost);	
//	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);	
//	cout<<"information "<<info<<endl;
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,a,d_At_l,r,d_Ipiv,d_At_r,r,d_info);
	hipDeviceSynchronize();
//	printTensor(d_At_r,r,a,1);
//now we obtain A' rewrite d_At_r and store as column
// we tanspose A' to A in d_A is a*r
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,a,r,&alpha,d_At_r,r,&beta,d_A,a,d_A,a);
//	cout<<"A"<<endl; printTensor(d_A,a,r,1);

// update B
//	cout<<"then updtate B"<<endl;
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,c,1,&alpha,d_A,a,a,d_C,c,c,&beta,d_CkrA,a,a*c,r);
//	cout<<"C kr A"<<endl;	printTensor(d_CkrA,c*a,r,1);
	f2h(d_CkrA,h_CkrA,a*c*r);
// compute (CkrA)'*X2', we have used batch matrix pro to get X2 
//	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,b,a*c,&alpha,d_CkrA,a*c,d_X2,b,&beta,d_Bt_r,r);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,b,a*c,&alpha,h_CkrA,HIP_R_16F,a*c,h_X2,HIP_R_16F,b,&beta,d_Bt_r,HIP_R_32F,r,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	cout<<"d_Bt_r"<<endl; printTensor(d_Bt_r,r,b,1);

	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,a,&alpha,d_A,a,d_A,a,&beta,d_ATA,r);
//	cout<<"A'A"<<endl;	printTensor(d_ATA,r,r,1);
	elepro<<<block1,threads>>>(d_CTC,d_ATA,d_Bt_l,r*r);
	hipDeviceSynchronize();
//	cout<<"Bt_l"<<endl; printTensor(d_Bt_l,r,r,1);
	//then we solve least squares minimization
	// (d_Bt_l)'B'=d_Bt_r ,due to d_Bt_l is symc so we don't tran, d_Bt_r has been  tran
	// r*r     r*b    r*b  store as col
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Bt_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Bt_l,r,d_work,d_Ipiv,d_info);
//	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);	
//	cout<<"information "<<info<<endl;
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,b,d_Bt_l,r,d_Ipiv,d_Bt_r,r,d_info);
	hipDeviceSynchronize();
//	printTensor(d_Bt_r,r,b,1);
//now we obtain B' rewrite d_Bt_r and store as column
// we tanspose B' to B in d_B is b*r
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,r,&alpha,d_Bt_r,r,&beta,d_B,b,d_B,b);
//	cout<<"B"<<endl; printTensor(d_B,b,r,1);

//update C
//	cout<<"update C"<<endl;
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,a*b,r);
//	cout<<"B kr A"<<endl;	printTensor(d_BkrA,b*a,r,1);
	f2h(d_BkrA,h_BkrA,a*b*r);
// compute (BkrA)'*X3', as Ct_r 
//	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,c,a*b,&alpha,d_BkrA,a*b,d_X,a*b,&beta,d_Ct_r,r);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,c,b*a,&alpha,h_BkrA,HIP_R_16F,b*a,h_X,HIP_R_16F,a*b,&beta,d_Ct_r,HIP_R_32F,r,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	cout<<"d_Ct_r"<<endl; printTensor(d_Ct_r,r,c,1);

	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);
//	cout<<"B'B"<<endl;	printTensor(d_BTB,r,r,1);
	elepro<<<block1,threads>>>(d_BTB,d_ATA,d_Ct_l,r*r);
	hipDeviceSynchronize();
//	cout<<"Ct_l"<<endl; printTensor(d_Ct_l,r,r,1);
	//then we solve least squares minimization
	// (d_Ct_l)'C'=d_Ct_r ,due to d_Ct_l is symc so we don't tran, d_Ct_r has been  tran
	// r*r     r*c    r*c  store as col
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Ct_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Ct_l,r,d_work,d_Ipiv,d_info);
//	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);	
//	cout<<"information "<<info<<endl;
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,c,d_Ct_l,r,d_Ipiv,d_Ct_r,r,d_info);
	hipDeviceSynchronize();
//	printTensor(d_Ct_r,r,c,1);
//now we obtain C' rewrite d_Ct_r and store as column
// we tanspose C' to C in d_C is c*r
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,r,&alpha,d_Ct_r,r,&beta,d_C,c,d_C,c);
//	cout<<"C"<<endl; printTensor(d_C,c,r,1);

/*	if(i == L-1){
	// recover to X3' which is same to X
	// X3'= (BkrA)*C' 
	dt *d_rec;
	hipMalloc((void**)&d_rec,sizeof(dt)*b*a*c);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r,&alpha,d_BkrA,a*b,d_C,c,&beta,d_rec,a*b);
	hipDeviceSynchronize();
	cout<<"d_X "<<endl; printTensor(d_X+100,4,4,1);
	cout<<"d_C "<<endl; printTensor(d_C+100,4,4,1);
	cout<<"d_BkrA "<<endl; printTensor(d_BkrA+100,4,4,1);
	cout<<"recover_rec "<<endl; printTensor(d_rec+100,4,4,1);
	// rec=-1*X+rec
//	hipblasSaxpy(handle,a*b*c,&alpha1,d_X,1,d_rec,1);
	elemin<<<block2,threads>>>(d_X,d_rec,a*b*c);
	cout<<"d_X "<<endl; printTensor(d_X+100,4,1,1);
	cout<<"rec-X "<<endl; printTensor(d_rec+100,4,1,1);
	//error rate = norm(res)/norm(X);
	hipblasSnrm2(handle,a*b*c,d_rec,1,&sh);
	cout<<"shang "<<endl; cout<<sh<<endl;
	hipblasSnrm2(handle,a*b*c,d_X,1,&xia);
	hipDeviceSynchronize();
	cout<<"xia "<<endl; cout<<xia<<endl;
	cout<<"error rate "<<sh/xia<<endl;
	hipFree(d_rec);
	}
*/
}
	hipFree(d_At_r);
	hipFree(d_At_l);
	hipFree(d_Bt_r);
	hipFree(d_Bt_l);
	hipFree(d_Ct_r);
	hipFree(d_Ct_l);

	hipMemcpyAsync(A,d_A,sizeof(dt)*a*r,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(B,d_B,sizeof(dt)*b*r,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(C,d_C,sizeof(dt)*c*r,hipMemcpyDeviceToHost,0);

	hipDeviceSynchronize();

	hipFree(h_X);hipFree(h_X2);
	hipFree(h_CkrB); hipFree(h_CkrA);hipFree(h_BkrA);
	
	hipFree(d_CkrB);
	hipFree(d_CkrA);
	hipFree(d_BkrA);

	hipFree(d_B);
	hipFree(d_X);
	hipFree(d_C);
	hipFree(d_A);
	hipFree(d_X2);

	hipFree(d_ATA);
	hipFree(d_BTB);
	hipFree(d_CTC);
//	hipHostFree(Ipiv);
	hipFree(d_Ipiv);
	hipFree(d_info);
	hipFree(d_work);
	hipsolverDnDestroy(cusolverH);
	hipblasDestroy(handle);
//	hipDeviceReset();
}
