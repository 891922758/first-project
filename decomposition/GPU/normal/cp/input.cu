#include "hip/hip_runtime.h"

#include "head.h"

int main(int argc,char *argv[]){

for(int i =160;i<=1300;i=i+160){
	clock_t t1,t2;
	double times=0.0;

	const long a = i;
	const long b = a;
	const long c = a;
	const long r = (int)(a*0.1);
	dt *X,*A,*B,*C;
	hipHostAlloc((void**)&X,sizeof(dt)*a*b*c,0);
	hipHostAlloc((void**)&A,sizeof(dt)*a*r,0);
	hipHostAlloc((void**)&B,sizeof(dt)*b*r,0);
	hipHostAlloc((void**)&C,sizeof(dt)*c*r,0);
	gencpTensor(X,a,b,c,r);

	t1=clock();
	cp_tensorcore(X,A,B,C,a,b,c,r);
	t2=clock();
	times = (double)(t2-t1)/CLOCKS_PER_SEC;
	cout<<i<<"   "<<times<<"s"<<endl;
/*
	// recover to X3' which is same to X
	// X3'= (BkrA)*C' 
	dt alpha = 1.0;
	dt beta = 0.0;
	dt sh=0.0;
	dt xia=1.0;
	dim3 threads(512,1,1);
	dim3 block1((r*r+512-1)/512,1,1); //for elepro
	dim3 block2((a*b*c+512-1)/512,1,1);
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt *d_rec,*d_B,*d_C,*d_A,*d_BkrA,*d_X;
	hipMalloc((void**)&d_rec,sizeof(dt)*b*a*c);
	hipMalloc((void**)&d_X,sizeof(dt)*b*a*c);
	hipMalloc((void**)&d_B,sizeof(dt)*b*r);
	hipMalloc((void**)&d_C,sizeof(dt)*c*r);
	hipMalloc((void**)&d_A,sizeof(dt)*a*r);
	hipMalloc((void**)&d_BkrA,sizeof(dt)*b*r*c);
	hipMemcpyAsync(d_X,X,sizeof(dt)*a*b*c,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_A,A,sizeof(dt)*a*r,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_B,B,sizeof(dt)*b*r,hipMemcpyHostToDevice,0);
	hipMemcpyAsync(d_C,C,sizeof(dt)*r*c,hipMemcpyHostToDevice,0);
	hipDeviceSynchronize();

	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,a*b,r);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r,&alpha,d_BkrA,a*b,d_C,c,&beta,d_rec,a*b);
	hipDeviceSynchronize();
	cout<<"d_X "<<endl; printTensor(d_X+100,4,4,1);
	cout<<"d_C "<<endl; printTensor(d_C+100,4,4,1);
	cout<<"d_BkrA "<<endl; printTensor(d_BkrA+100,4,4,1);
	cout<<"recover_rec "<<endl; printTensor(d_rec+100,4,4,1);
	// rec=-1*X+rec
//	hipblasSaxpy(handle,a*b*c,&alpha1,d_X,1,d_rec,1);
	elemin<<<block2,threads>>>(d_X,d_rec,a*b*c);
	cout<<"d_X "<<endl; printTensor(d_X+100,4,1,1);
	cout<<"rec-X "<<endl; printTensor(d_rec+100,4,1,1);
	//error rate = norm(res)/norm(X);
	hipblasSnrm2(handle,a*b*c,d_rec,1,&sh);
	cout<<"shang "<<endl; cout<<sh<<endl;
	hipblasSnrm2(handle,a*b*c,d_X,1,&xia);
	hipDeviceSynchronize();
	cout<<"xia "<<endl; cout<<xia<<endl;
	cout<<"error rate "<<sh/xia<<endl;
	hipFree(d_rec);
	hipFree(d_B);
	hipFree(d_A);
	hipFree(d_C);
	hipFree(d_BkrA);
	hipFree(d_X);
*/
	ofstream outfile("cptensorcore.txt",ios::app);
	outfile<<i<<"   "<<times<<"s"<<endl;
	outfile.close();

	hipHostFree(X);
	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

}
	return 0;
}

