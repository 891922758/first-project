#include "hip/hip_runtime.h"
#include "head.h"

void tucker_hosvd(dt *X,dt *core,dt *U1,dt *U2,dt *U3,long a,long b,long c,long r1,long r2,long r3){
	//X is a*b*c, core is r1*r2*r3, U1 is a*r1,U2 b*r2,U3 is c*r3 
	const int mp = 6;  //max divide 6 parts
	int p,slice,le;  // each part process slice matrices, leave le matrix
	if(c%5==0){
		p = 5;   //5 parts
		slice = c/5; 
	}else{
		p = 6;
		slice = c/5;
		le = c%5;
	}

//	p = 2;
//	slice = 1;
	dim3 threads(512,1,1);
	dim3 block0((a*a+512-1)/512,1,1); //for X2
	dim3 block1((slice*slice+512-1)/512,1,1); //for X3

	dt alpha = 1.0;
	dt alpha1 = -1.0;
	dt beta = 0.0;
	dt beta1 = 1.0;
	dt sh=0.0;
	dt xia=1.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	dt *d_work = NULL;
	int lwork=0;
	int *devInfo=NULL;
	hipMalloc((void**)&devInfo,sizeof(int));
	int infogpu=0;

	dt *d_X1_X1,*d_X2_X2,*d_X3_X3;
	dt  *d_core;
	hipMalloc((void**)&d_core,sizeof(dt)*r1*r2*r3);
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);

	hipDeviceSynchronize();
	//cout<<"X"<<endl; printTensor(d_X,a,b,c);
if(p==5){
	dt *d_Idemat,*d_X2,*d_X3,*d_X3T;
	dt *d_Idemat3;
	hipMalloc((void**)&d_Idemat3,sizeof(dt)*slice*slice);
	hipMalloc((void**)&d_Idemat,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*slice);
	hipMalloc((void**)&d_X3,sizeof(dt)*c*c);
	hipMalloc((void**)&d_X3T,sizeof(dt)*c*slice);
	initIdeMat<<<block0,threads>>>(d_Idemat,a);
	initIdeMat<<<block1,threads>>>(d_Idemat3,slice);

	dt *d_Xtemp,*d_Xtemp1;
	hipMalloc((void**)&d_Xtemp,sizeof(dt)*a*b*slice);
	hipMalloc((void**)&d_Xtemp1,sizeof(dt)*a*b*slice);
	for(int i = 0;i<p;i++){
		hipMemcpyAsync(d_Xtemp,X+i*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice,0);
		hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b*slice,&alpha,d_Xtemp,a,d_Xtemp,a,&beta1,d_X1_X1,a);
		hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,a,a,&alpha,d_Xtemp,a,a*b,d_Idemat,a,0,&beta,d_X2,b,b*a,slice);
		hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,b,a,&alpha,d_X2,b,d_X2,b,&beta1,d_X2_X2,b);
		for (int j = 0;j<p;j++){
			hipMemcpyAsync(d_Xtemp1,X+j*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice,0);
			hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,slice,slice,a*b,&alpha,d_Xtemp1,a*b,d_Xtemp,a*b,&beta,d_X3+(i*p+j)*slice*slice,slice);
		//	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,1,a*b,&alpha,d_Xtemp,1,d_Xtemp1,a*b,&beta,d_X3_X3+i*c+j,1);
		}// d_X3 is size of slice *c transpose to c*slice
		hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,slice,slice,slice,&alpha,d_X3+i*c*slice,slice,slice*slice,d_Idemat3,slice,0,&beta,d_X3T,slice,slice*slice,p);
		hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,slice,&alpha,d_X3T,slice,&beta,d_X3_X3+i*c*slice,c,d_X3_X3+i*c*slice,c);
	}
//	cout<<"X3X3"<<endl; printTensor(d_X3_X3,c,c,1);
	hipFree(d_Xtemp1);
	hipFree(d_Idemat);
	hipFree(d_X2);
	hipFree(d_X3);
	hipFree(d_X3T);
	hipFree(d_Idemat3);
	hipDeviceSynchronize();

	// syevd for U1,U2,U3
	//data prepare for store eigvalue and eigvectors,we only fetch r1 r2 and r3 eigvectors from origin
	dt *d_W1,*d_W2,*d_W3; 
	hipMalloc((void**)&d_W1,sizeof(dt)*a);
	hipMalloc((void**)&d_W2,sizeof(dt)*b);
	hipMalloc((void**)&d_W3,sizeof(dt)*c);
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverEigRange_t range = HIPSOLVER_EIG_RANGE_ALL;
	int meig1=a; int meig2=b; int meig3=c;
	// turn X1X1 to eigvectores and view as U1
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,d_work,lwork,devInfo);
//	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<infogpu<<endl;
	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"first vectores"<<endl; printTensor(d_X1_X1+(a-r1)*a,a,r1,1);

	// turn X2X2 to eigvectores and view as U2
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,d_work,lwork,devInfo);
	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<infogpu<<endl;
//	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"second vectors"<<endl; printTensor(d_X2_X2+(b-r2)*b,b,r2,1);
	
	// turn X3X3 to eigvectores and view as U3
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,d_work,lwork,devInfo);
//	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<infogpu<<endl;
	hipDeviceSynchronize();
	//cout<<"third vectores"<<endl; printTensor(d_X3_X3+(c-r3)*c,c,r3,1);
	hipFree(d_W1);
	hipFree(d_W2);
	hipFree(d_W3);

	// then compute X x1U1 x2U2 x3U3,we need extra two intenal vals and core to store last result 
	// a*b*c  a*r1  b*r2  c*r3
	// X x1U1' =U1'*X1  X1 can obtain direct store as X
	dt *d_XU1,*d_XU1U2;
	hipMalloc((void**)&d_XU1,sizeof(dt)*r1*b*slice);
	hipMalloc((void**)&d_XU1U2,sizeof(dt)*r1*r2*c);
	for(int i = 0;i<p;i++){
		hipMemcpyAsync(d_Xtemp,X+i*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice,0);
		hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r1,b*slice,a,&alpha,d_X1_X1+(a-r1)*a,a,d_Xtemp,a,&beta,d_XU1,r1);
		hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,d_XU1,r1,r1*b,d_X2_X2+(b-r2)*b,b,0,&beta,d_XU1U2+i*r1*r2*slice,r1,r1*r2,slice);
	//	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,d_XU1,r1,d_X2_X2+(b-r2)*b,b,&beta,d_XU1U2+i*r1*r2,r1);
	
	}
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2,r3,c,&alpha,d_XU1U2,r1*r2,d_X3_X3+(c-r3)*c,c,&beta,d_core,r1*r2);
	hipFree(d_XU1);
	hipFree(d_XU1U2);
	hipFree(d_Xtemp);
	hipDeviceSynchronize();

}

	//recover by X = core X1U1 X2U2 X3U3
	// r1*r2*r3  a*r1 ,b*r2 ,c*r3
/*	dt *d_X;
	hipMalloc((void**)&d_X,sizeof(dt)*a*b*c);
	hipMemcpyAsync(d_X,X,sizeof(dt)*a*b*c,hipMemcpyHostToDevice,0);

	dt *d_coreU1,*d_coreU1U2,*d_rec;
	hipMalloc((void**)&d_coreU1,sizeof(dt)*a*r2*r3);
	hipMalloc((void**)&d_coreU1U2,sizeof(dt)*a*b*r3);
	hipMalloc((void**)&d_rec,sizeof(dt)*a*b*c);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a,r2*r3,r1,&alpha,d_X1_X1+(a-r1)*a,a,d_core,r1,&beta,d_coreU1,a);
	//cout<<"coreU1"<<endl; printTensor(d_coreU1,a,r2,r3);
	//a*r2*r3  b*r2 coreU1U2
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,r2,&alpha,d_coreU1,a,a*r2,d_X2_X2+(b-r2)*b,b,0,&beta,d_coreU1U2,a,a*b,r3);
	//cout<<"coreU1U2"<<endl; printTensor(d_coreU1U2,a,b,r3);
	//a*b*r3 c*r3 rec a*b*c
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r3,&alpha,d_coreU1U2,a*b,d_X3_X3+(c-r3)*c,c,&beta,d_rec,a*b);
//	cout<<"rec"<<endl; printTensor(d_rec,a,b,c);
	hipblasSaxpy(handle,a*b*c,&alpha1,d_X,1,d_rec,1);
//	cout<<"rec"<<endl; printTensor(d_rec,a,b,c);
	hipblasSnrm2(handle,a*b*c,d_rec,1,&sh);
	cout<<"sh "<<endl; cout<<sh<<endl;
	hipblasSnrm2(handle,a*b*c,d_X,1,&xia);
	hipDeviceSynchronize();
	cout<<"xi "<<endl; cout<<xia<<endl;
	cout<<"error rate "<<sh/xia<<endl;

	hipDeviceSynchronize();
	hipFree(d_X);
	hipFree(d_coreU1);
	hipFree(d_coreU1U2);
	hipFree(d_rec);
*/

	// transfer result to Host
	hipMemcpyAsync(core,d_core,sizeof(dt)*r1*r2*r3,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(U1,d_X1_X1+(a-r1)*a,sizeof(dt)*a*r1,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(U2,d_X2_X2+(b-r2)*b,sizeof(dt)*b*r2,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(U3,d_X3_X3+(c-r3)*c,sizeof(dt)*c*r3,hipMemcpyDeviceToHost,0);
//	printTensor(d_X1_X1+(a-r1)*a,a,r1,1);
//	printTensor(d_X2_X2+(b-r2)*b,b,r2,1);
//	printTensor(d_X3_X3+(c-r3)*a,c,r3,1);
//	printTensor(d_core,r1,r2,r3);

	hipDeviceSynchronize();
	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
	hipFree(d_core);
	hipFree(d_work);
	hipFree(devInfo);
	hipFree(d_X1_X1);
	hipFree(d_X2_X2);
	hipFree(d_X3_X3);
	hipDeviceReset();
}

