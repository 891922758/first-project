#include "hip/hip_runtime.h"
#include "head.h"

void cp_als(dt *X,dt *A,dt *B,dt *C,long a,long b,long c,long r){
// X is a*b*c; A is a*r; B is b*r; C is c*r
// we assume they all store as column
//	const int mp = 6;  //max divide 6 parts
	int p,slice,le;  // each part process slice matrices, leave le matrix
	if(c%5==0){
		p = 5;         //5 parts
		slice = c/5; 
	}else{
		p = 6;
		slice = c/5;
		le = c%5;
	}

	dt alpha = 1.0;
	dt beta = 0.0;
	dt beta1 = 1.0;
//	dt sh=0.0;
//	dt xia=1.0;
	dim3 threads(512,1,1);
	dim3 block1((r*r+512-1)/512,1,1); //for elepro
	dim3 block0((a*a+512-1)/512,1,1); //for X2
	dim3 block2((a*b*c+512-1)/512,1,1); 
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	int info = 0;
	int *d_info = NULL;
	hipMalloc((void**)&d_info,sizeof(int));
//	int *Ipiv; 
//	hipHostAlloc((void**)&Ipiv,sizeof(int)*r,0);
	int *d_Ipiv = NULL; // PA=LU, P is control weather permute
	hipMalloc((void**)&d_Ipiv,sizeof(int));
	int lwork=0;
	dt *d_work = NULL;

	dt *d_C,*d_B,*d_A;
	hipMalloc((void**)&d_B,sizeof(dt)*b*r);
	hipMalloc((void**)&d_C,sizeof(dt)*c*r);
	hipMalloc((void**)&d_A,sizeof(dt)*a*r);

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	hiprandGenerateUniform(gen,d_B,b*r);
	hiprandGenerateUniform(gen,d_C,c*r);

//	hipMemcpyAsync(d_B,B,sizeof(dt)*b*r,hipMemcpyHostToDevice,0);
//	hipMemcpyAsync(d_C,C,sizeof(dt)*c*r,hipMemcpyHostToDevice,0);
//	cout<<"B"<<endl; printTensor(d_B,b,r,1);
//	cout<<"C"<<endl; printTensor(d_C,c,r,1);
//	cout<<"X"<<endl; printTensor(d_X,a,b,c);

	dt *d_ATA,*d_BTB,*d_CTC;
	hipMalloc((void**)&d_ATA,sizeof(dt)*r*r);
	hipMalloc((void**)&d_BTB,sizeof(dt)*r*r);
	hipMalloc((void**)&d_CTC,sizeof(dt)*r*r);
	dt *d_tempABC;  //for all temp
	hipMalloc((void**)&d_tempABC,sizeof(dt)*a*b*slice);
	dt *d_tempB;  // for X2 solve B
	hipMalloc((void**)&d_tempB,sizeof(dt)*a*b*slice);

	dt *d_CkrB;
	hipMalloc((void**)&d_CkrB,sizeof(dt)*b*c*r); //GPU store C kr B
	dt *d_CkrA;
	hipMalloc((void**)&d_CkrA,sizeof(dt)*a*c*r); //GPU store C kr A
	dt *d_BkrA;
	hipMalloc((void**)&d_BkrA,sizeof(dt)*a*b*r); //GPU store B kr A
	dt *d_CkrBt;
	hipMalloc((void**)&d_CkrBt,sizeof(dt)*b*c*r); //GPU store C kr B
	dt *d_CkrAt;
	hipMalloc((void**)&d_CkrAt,sizeof(dt)*a*c*r); //GPU store C kr A

	const int L = 10;
if(p==5){
	dt *d_Idemat;
	hipMalloc((void**)&d_Idemat,sizeof(dt)*a*a);
	initIdeMat<<<block0,threads>>>(d_Idemat,a);

 for(int i = 0;i<L;i++){
// update A
//	cout<<"unpdta A"<<endl;
// we compute kr(dot) product
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,c,1,&alpha,d_B,b,b,d_C,c,c,&beta,d_CkrB,b,b*c,r);
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,b*c,&alpha,d_CkrB,b*c,&beta,d_CkrBt,r,d_CkrBt,r);
//cout<<"CkrB"<<endl; printTensor(d_CkrB,c*b,r,1);
// compute (CkrB)'*X1'
	dt *d_At_r;
	hipMalloc((void**)&d_At_r,sizeof(dt)*a*r); //GPU store (CkrB)'*X1' as right part 
	dt *d_At_l;
 	hipMalloc((void**)&d_At_l,sizeof(dt)*r*r); //GPU store (CTC.*BTB)' as left part
   for(int j= 0;j<p;j++){

//	hipMemcpy(d_tempABC,X+j*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice);
	hipMemcpyAsync(d_tempABC,X+j*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice,0);
	hipDeviceSynchronize();
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,r,a,b*slice,&alpha,d_CkrBt+j*r*b*slice,r,d_tempABC,a,&beta1,d_At_r,r);
}
//cout<<"CkrB'*X1'"<<endl;printTensor(d_At_r,r,a,1);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,c,&alpha,d_C,c,d_C,c,&beta,d_CTC,r);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);
//cout<<"CTC"<<endl; printTensor(d_CTC,r,r,1);
//cout<<"BTB"<<endl; printTensor(d_BTB,r,r,1);
	// compute (CTC.*BTB)'  
	elepro<<<block1,threads>>>(d_CTC,d_BTB,d_At_l,r*r);
//cout<<"CTC.*BTB"<<endl;printTensor(d_At_l,r,r,1);

	//then we solve least squares minimization
	// (d_At_l)'A'=d_At_r ,due to d_At_l is symc so we don't tran, d_At_r has implity tran
	// r*r     r*a    r*a store as col
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_At_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_At_l,r,d_work,d_Ipiv,d_info);
//	hipMemcpy(Ipiv,d_Ipiv,sizeof(int)*r,hipMemcpyDeviceToHost);	
//	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);	
//	cout<<"information "<<info<<endl;
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,a,d_At_l,r,d_Ipiv,d_At_r,r,d_info);
	hipDeviceSynchronize();
//	printTensor(d_At_r,r,a,1);
//now we obtain A' rewrite d_At_r and store as column
// we tanspose A' to A in d_A is a*r
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,a,r,&alpha,d_At_r,r,&beta,d_A,a,d_A,a);
//cout<<"A"<<endl; printTensor(d_A,a,r,1);

// update B
//	cout<<"then updtate B"<<endl;
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,c,1,&alpha,d_A,a,a,d_C,c,c,&beta,d_CkrA,a,a*c,r);
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,a*c,&alpha,d_CkrA,a*c,&beta,d_CkrAt,r,d_CkrAt,r);
//	cout<<"C kr A"<<endl;	printTensor(d_CkrA,c*a,r,1);
// compute (CkrA)'*X2', we have used batch matrix pro to get X2 
	dt *d_Bt_r;
	hipMalloc((void**)&d_Bt_r,sizeof(dt)*b*r); //GPU store (CkrA)'*X2' as right part 
for(int j =0;j<p;j++){
	hipMemcpy(d_tempABC,X+j*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,a,a,&alpha,d_tempABC,a,a*b,d_Idemat,a,0,&beta,d_tempB,b,b*a,slice);
	
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,r,b,a*slice,&alpha,d_CkrAt+j*r*a*slice,r,d_tempB,b,&beta1,d_Bt_r,r);
	
}
//	cout<<"d_Bt_r"<<endl; printTensor(d_Bt_r,r,b,1);

	dt *d_Bt_l;
	hipMalloc((void**)&d_Bt_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,a,&alpha,d_A,a,d_A,a,&beta,d_ATA,r);
//	cout<<"A'A"<<endl;	printTensor(d_ATA,r,r,1);
	elepro<<<block1,threads>>>(d_CTC,d_ATA,d_Bt_l,r*r);
	hipDeviceSynchronize();
//	cout<<"Bt_l"<<endl; printTensor(d_Bt_l,r,r,1);
	//then we solve least squares minimization
	// (d_Bt_l)'B'=d_Bt_r ,due to d_Bt_l is symc so we don't tran, d_Bt_r has been  tran
	// r*r     r*b    r*b  store as col
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Bt_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Bt_l,r,d_work,d_Ipiv,d_info);
//	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);	
//	cout<<"information "<<info<<endl;
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,b,d_Bt_l,r,d_Ipiv,d_Bt_r,r,d_info);
	hipDeviceSynchronize();
//	printTensor(d_Bt_r,r,b,1);
//now we obtain B' rewrite d_Bt_r and store as column
// we tanspose B' to B in d_B is b*r
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,r,&alpha,d_Bt_r,r,&beta,d_B,b,d_B,b);
//	cout<<"B"<<endl; printTensor(d_B,b,r,1);

//update C
//	cout<<"update C"<<endl;
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,a*b,r);
//	cout<<"B kr A"<<endl;	printTensor(d_BkrA,b*a,r,1);
// compute (BkrA)'*X3', as Ct_r 
	dt *d_Ct_r;
	hipMalloc((void**)&d_Ct_r,sizeof(dt)*c*r); //GPU store (BkrA)'*X3' as right part 
for(int j=0;j<p;j++){
	hipMemcpy(d_tempABC,X+j*a*b*slice,sizeof(dt)*a*b*slice,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,slice,a*b,&alpha,d_BkrA,a*b,d_tempABC,a*b,&beta,d_Ct_r+j*r*slice,r);
}
//	cout<<"d_Ct_r"<<endl; printTensor(d_Ct_r,r,c,1);

	dt *d_Ct_l;
	hipMalloc((void**)&d_Ct_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);
//	cout<<"B'B"<<endl;	printTensor(d_BTB,r,r,1);
	elepro<<<block1,threads>>>(d_BTB,d_ATA,d_Ct_l,r*r);
	hipDeviceSynchronize();
//	cout<<"Ct_l"<<endl; printTensor(d_Ct_l,r,r,1);
	//then we solve least squares minimization
	// (d_Ct_l)'C'=d_Ct_r ,due to d_Ct_l is symc so we don't tran, d_Ct_r has been  tran
	// r*r     r*c    r*c  store as col
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Ct_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Ct_l,r,d_work,d_Ipiv,d_info);
//	hipMemcpy(&info,d_info,sizeof(int),hipMemcpyDeviceToHost);	
//	cout<<"information "<<info<<endl;
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,c,d_Ct_l,r,d_Ipiv,d_Ct_r,r,d_info);
	hipDeviceSynchronize();
//	printTensor(d_Ct_r,r,c,1);
//now we obtain C' rewrite d_Ct_r and store as column
// we tanspose C' to C in d_C is c*r
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,r,&alpha,d_Ct_r,r,&beta,d_C,c,d_C,c);
//	cout<<"C"<<endl; printTensor(d_C,c,r,1);
  
/*    if(i==L-1){	
	// recover to X3' which is same to X
	// X3'= (BkrA)*C' 
	dt *d_rec;
	hipMalloc((void**)&d_rec,sizeof(dt)*b*a*c);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r,&alpha,d_BkrA,a*b,d_C,c,&beta,d_rec,a*b);
//	cout<<"recover "<<endl; printTensor(d_rec,a*b,c,1);
	// rec=-1*X+rec
//	hipblasSaxpy(handle,a*b*c,&alpha1,d_X,1,d_rec,1);
	elemin<<<block2,threads>>>(d_X,d_rec,a*b*c);
	//error rate = norm(res)/norm(X);
	hipblasSnrm2(handle,a*b*c,d_rec,1,&sh);
	cout<<"shang "<<endl; cout<<sh<<endl;
	hipblasSnrm2(handle,a*b*c,d_X,1,&xia);
	hipDeviceSynchronize();
	cout<<"xia "<<endl; cout<<xia<<endl;
	cout<<"error rate "<<sh/xia<<endl;
	hipFree(d_rec);
     }
*/
	hipFree(d_At_r);
	hipFree(d_At_l);
	hipFree(d_Bt_r);
	hipFree(d_Bt_l);
	hipFree(d_Ct_r);
	hipFree(d_Ct_l);
}
	hipFree(d_Idemat);

}
	hipMemcpyAsync(A,d_A,sizeof(dt)*a*r,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(B,d_B,sizeof(dt)*b*r,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(C,d_C,sizeof(dt)*c*r,hipMemcpyDeviceToHost,0);

//	printTensor(d_A,a,r,1);
//	printTensor(d_B,b,r,1);
//	printTensor(d_C,c,r,1);

	hipDeviceSynchronize();
	hipFree(d_CkrB);
	hipFree(d_CkrA);
	hipFree(d_BkrA);
	hipFree(d_CkrBt);
	hipFree(d_CkrAt);
	hipFree(d_tempABC);
	hipFree(d_tempB);

	hipFree(d_B);
	hipFree(d_C);
	hipFree(d_A);

	hipFree(d_ATA);
	hipFree(d_BTB);
	hipFree(d_CTC);
	hipFree(d_Ipiv);
	hipFree(d_info);
	hipFree(d_work);
	hiprandDestroyGenerator(gen);
	hipsolverDnDestroy(cusolverH);
	hipblasDestroy(handle);
	hipDeviceReset();
}
