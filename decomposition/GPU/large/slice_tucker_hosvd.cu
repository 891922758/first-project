#include "head.h"

void tucker_hosvd(dt *X,dt *core,dt *U1,dt *U2,dt *U3,long a,long b,long c,long r1,long r2,long r3){
	//X is a*b*c, core is r1*r2*r3, U1 is a*r1,U2 b*r2,U3 is c*r3 

	dt alpha = 1.0;
	dt alpha1 = -1.0;
	dt beta = 0.0;
	dt beta1 = 1.0;
	dt sh=0.0;
	dt xia=1.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	dt *d_work = NULL;
	int lwork=0;
	int *devInfo=NULL;
	hipMalloc((void**)&devInfo,sizeof(int));
	int infogpu=0;

	dt *d_X1_X1,*d_X2_X2,*d_X3_X3;
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);
	dt *d_Xtemp,*d_Xtemp1;
	hipMalloc((void**)&d_Xtemp,sizeof(dt)*a*b);
	hipMalloc((void**)&d_Xtemp1,sizeof(dt)*a*b);

	hipDeviceSynchronize();
	//cout<<"X"<<endl; printTensor(d_X,a,b,c);

	//compute X1*X1' a*bc * bc*a
for(int i = 0;i<c;i++){
	hipMemcpyAsync(d_Xtemp,X+i*a*b,sizeof(dt)*a*b,hipMemcpyHostToDevice,0);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b,&alpha,d_Xtemp,a,d_Xtemp,a,&beta1,d_X1_X1,a);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,b,a,&alpha,d_Xtemp,a,d_Xtemp,a,&beta1,d_X2_X2,b);
	for (int j = 0;j<c;j++){
		hipMemcpyAsync(d_Xtemp1,X+j*a*b,sizeof(dt)*a*b,hipMemcpyHostToDevice,0);
		hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,1,1,a*b,&alpha,d_Xtemp,1,d_Xtemp1,a*b,&beta,d_X3_X3+i*c+j,1);
		
	}
}
	cout<<"d_X3X3"<<endl; printTensor(d_X3_X3,c,c,1);
	hipFree(d_Xtemp1);
	hipDeviceSynchronize();

	// syevd for U1,U2,U3
	//data prepare for store eigvalue and eigvectors,we only fetch r1 r2 and r3 eigvectors from origin
	dt *d_W1,*d_W2,*d_W3; 
	hipMalloc((void**)&d_W1,sizeof(dt)*a);
	hipMalloc((void**)&d_W2,sizeof(dt)*b);
	hipMalloc((void**)&d_W3,sizeof(dt)*c);
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverEigRange_t range = HIPSOLVER_EIG_RANGE_ALL;
	int meig1=a; int meig2=b; int meig3=c;
	// turn X1X1 to eigvectores and view as U1
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,d_work,lwork,devInfo);
	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
	cout<<infogpu<<endl;
	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"first vectores"<<endl; printTensor(d_X1_X1+(a-r1)*a,a,r1,1);

	// turn X2X2 to eigvectores and view as U2
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,d_work,lwork,devInfo);
	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
	cout<<infogpu<<endl;
	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"second vectors"<<endl; printTensor(d_X2_X2+(b-r2)*b,b,r2,1);
	
	// turn X3X3 to eigvectores and view as U3
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,d_work,lwork,devInfo);
	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
	cout<<infogpu<<endl;
	hipDeviceSynchronize();
	//cout<<"third vectores"<<endl; printTensor(d_X3_X3+(c-r3)*c,c,r3,1);
	hipFree(d_W1);
	hipFree(d_W2);
	hipFree(d_W3);

	// then compute X x1U1 x2U2 x3U3,we need extra two intenal vals and core to store last result 
	// a*b*c  a*r1  b*r2  c*r3
	// X x1U1' =U1'*X1  X1 can obtain direct store as X
	dt *d_XU1,*d_XU1U2;
	  dt  *d_core;
	hipMalloc((void**)&d_XU1,sizeof(dt)*r1*b);
	hipMalloc((void**)&d_XU1U2,sizeof(dt)*r1*r2*c);
	hipMalloc((void**)&d_core,sizeof(dt)*r1*r2*r3);
for(int i = 0;i<c;i++){
	hipMemcpyAsync(d_Xtemp,X+i*a*b,sizeof(dt)*a*b,hipMemcpyHostToDevice,0);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r1,b,a,&alpha,d_X1_X1+(a-r1)*a,a,d_Xtemp,a,&beta,d_XU1,r1);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,d_XU1,r1,d_X2_X2+(b-r2)*b,b,&beta,d_XU1U2+i*r1*r2,r1);
	
}
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2,r3,c,&alpha,d_XU1U2,r1*r2,d_X3_X3+(c-r3)*c,c,&beta,d_core,r1*r2);
	hipFree(d_XU1);
	hipFree(d_XU1U2);
	hipFree(d_Xtemp);
	hipDeviceSynchronize();

	//X X1 U1 a*b*c a*r1
	//hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r1,b*c,a,&alpha,d_X1_X1+(a-r1)*a,a,d_X,a,&beta,d_XU1,r1);
	//cout<<"XU1"<<endl; printTensor(d_XU1,r1,b*c,1);
	//XU1*U2 r1*b *c  b*r2
	//hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,d_XU1,r1,r1*b,d_X2_X2+(b-r2)*b,b,0,&beta,d_XU1U2,r1,r1*r2,c);
	//cout<<"XU1U2"<<endl; printTensor(d_XU1U2,r1,r2*c,1);
	//XU1U2*U3'  r1*r2*r3 c*r3
//	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2,r3,c,&alpha,d_XU1U2,r1*r2,d_X3_X3+(c-r3)*c,c,&beta,d_core,r1*r2);
	//cout<<"core"<<endl; printTensor(d_core,1,r1*r2*r3,1);

	//recover by X = core X1U1 X2U2 X3U3
	// r1*r2*r3  a*r1 ,b*r2 ,c*r3
/*	dt *d_X;
	hipMalloc((void**)&d_X,sizeof(dt)*a*b*c);
	hipMemcpyAsync(d_X,X,sizeof(dt)*a*b*c,hipMemcpyHostToDevice,0);

	dt *d_coreU1,*d_coreU1U2,*d_rec;
	hipMalloc((void**)&d_coreU1,sizeof(dt)*a*r2*r3);
	hipMalloc((void**)&d_coreU1U2,sizeof(dt)*a*b*r3);
	hipMalloc((void**)&d_rec,sizeof(dt)*a*b*c);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a,r2*r3,r1,&alpha,d_X1_X1+(a-r1)*a,a,d_core,r1,&beta,d_coreU1,a);
	//cout<<"coreU1"<<endl; printTensor(d_coreU1,a,r2,r3);
	//a*r2*r3  b*r2 coreU1U2
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,r2,&alpha,d_coreU1,a,a*r2,d_X2_X2+(b-r2)*b,b,0,&beta,d_coreU1U2,a,a*b,r3);
	//cout<<"coreU1U2"<<endl; printTensor(d_coreU1U2,a,b,r3);
	//a*b*r3 c*r3 rec a*b*c
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r3,&alpha,d_coreU1U2,a*b,d_X3_X3+(c-r3)*c,c,&beta,d_rec,a*b);
//	cout<<"rec"<<endl; printTensor(d_rec,a,b,c);
	hipblasSaxpy(handle,a*b*c,&alpha1,d_X,1,d_rec,1);
//	cout<<"rec"<<endl; printTensor(d_rec,a,b,c);
	hipblasSnrm2(handle,a*b*c,d_rec,1,&sh);
	cout<<"sh "<<endl; cout<<sh<<endl;
	hipblasSnrm2(handle,a*b*c,d_X,1,&xia);
	hipDeviceSynchronize();
	cout<<"xi "<<endl; cout<<xia<<endl;
	cout<<"error rate "<<sh/xia<<endl;

	hipDeviceSynchronize();
	hipFree(d_X);
	hipFree(d_coreU1);
	hipFree(d_coreU1U2);
	hipFree(d_rec);
*/

	// transfer result to Host
	hipMemcpyAsync(core,d_core,sizeof(dt)*r1*r2*r3,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(U1,d_X1_X1+(a-r1)*a,sizeof(dt)*a*r1,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(U2,d_X2_X2+(b-r2)*b,sizeof(dt)*b*r2,hipMemcpyDeviceToHost,0);
	hipMemcpyAsync(U3,d_X3_X3+(c-r3)*c,sizeof(dt)*c*r3,hipMemcpyDeviceToHost,0);
	printTensor(d_X1_X1+(a-r1)*a,a,r1,1);
	printTensor(d_X2_X2+(b-r2)*b,b,r2,1);
	printTensor(d_X3_X3+(c-r3)*a,c,r3,1);
	printTensor(d_core,r1,r2,r3);

	hipDeviceSynchronize();
	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
	hipFree(d_core);
	hipFree(d_work);
	hipFree(devInfo);
	hipFree(d_X1_X1);
	hipFree(d_X2_X2);
	hipFree(d_X3_X3);
	hipDeviceReset();
}

