
#include "cudecompose.h"

void rse(dt *d_X,dt *d_X1,long m,dt *error){
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt sh = 0.0;
	dt xia = 1.0;
	tminust(d_X,d_X1,m);
	hipblasSnrm2(handle,m,d_X1,1,&sh);
	hipblasSnrm2(handle,m,d_X,1,&xia);
	*error = sh/xia;
	hipDeviceSynchronize();
	hipblasDestroy(handle);
}

void gencp(dt *d_rec,dt *d_A,dt *d_B,dt *d_C,long a,long b,long c,long r){
	// rec = A*CkrB'  a*bc = a*r  bc*r
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;
	dt *d_CkrB;
	hipMalloc((void**)&d_CkrB,sizeof(dt)*b*c*r);
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,c,1,&alpha,d_B,b,b,d_C,c,c,&beta,d_CkrB,b,b*c,r);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b*c,r,&alpha,d_A,a,d_CkrB,b*c,&beta,d_rec,a);
	hipDeviceSynchronize();
	hipFree(d_CkrB);
	hipblasDestroy(handle);
}

void gentucker(dt *d_rec,dt *d_core, dt *d_A,dt *d_B,dt *d_C,long a,long b,long c,long r1,long r2,long r3){
	// rec = core x1 A x2 B X3 C
	// a*b*c  r1* r2 *r3  a*r1 b*r2 c*r3
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;
	dt *d_coreA,*d_coreAB;
	hipMalloc((void**)&d_coreA,sizeof(dt)*a*r2*r3);
	hipMalloc((void**)&d_coreAB,sizeof(dt)*a*b*r3);

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a,r2*r3,r1,&alpha,d_A,a,d_core,r1,&beta,d_coreA,a);
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,r2,&alpha,d_coreA,a,a*r2,d_B,b,0,&beta,d_coreAB,a,a*b,r3);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r3,&alpha,d_coreAB,a*b,d_C,c,&beta,d_rec,a*b);

	hipDeviceSynchronize();
	hipFree(d_coreA);
	hipFree(d_coreAB);
	hipblasDestroy(handle);
}

void gencp4(dt *d_T,dt *d_AA,dt *d_BB,dt *d_CC,dt *d_DD,long a,long b,long c,long d,long r){
//X(1) = A*(DkrCkrB)' a*r r*bcd 
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;

	dt *d_CKRB,*d_kr;
	hipMalloc((void**)&d_CKRB,sizeof(dt)*c*r*b);
	hipMalloc((void**)&d_kr,sizeof(dt)*c*r*b*d);
	hipDeviceSynchronize();

	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,c,1,&alpha,d_BB,b,b,d_CC,c,c,&beta,d_CKRB,b,b*c,r);
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b*c,d,1,&alpha,d_CKRB,b*c,b*c,d_DD,d,d,&beta,d_kr,b*c,b*c*d,r);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b*c*d,r,&alpha,d_AA,a,d_kr,b*c*d,&beta,d_T,a);
	hipDeviceSynchronize();

	hipFree(d_CKRB);
	hipFree(d_kr);
	hipblasDestroy(handle);
}

void gentucker4(dt *d_T,dt *d_G, dt *d_A,dt *d_B,dt *d_C,dt *d_D,long a,long b,long c,long d,long r1,long r2,long r3,long r4){
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	dt alpha = 1.0;
	dt beta = 0.0;
	dt *d_coreU1,*d_coreU1U2,*d_coreU1U2U3;
	hipMalloc((void**)&d_coreU1,sizeof(dt)*a*r2*r3*r4);
	hipMalloc((void**)&d_coreU1U2,sizeof(dt)*a*b*r3*r4);
	hipMalloc((void**)&d_coreU1U2U3,sizeof(dt)*a*b*c*r4);

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,a,r2*r3*r4,r1,&alpha,d_A,a,d_G,r1,&beta,d_coreU1,a);
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,r2,&alpha,d_coreU1,a,a*r2,d_B,b,0,&beta,d_coreU1U2,a,a*b,r3*r4);
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b,c,r3,&alpha,d_coreU1U2,a*b,a*b*r3,d_C,c,0,&beta,d_coreU1U2U3,a*b,a*b*c,r4);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a*b*c,d,r4,&alpha,d_coreU1U2U3,a*b*c,d_D,d,&beta,d_T,a*b*c);
	hipDeviceSynchronize();

	hipFree(d_coreU1);
	hipFree(d_coreU1U2);
	hipFree(d_coreU1U2U3);

	hipblasDestroy(handle);
}

