#include "cudecompose.h"

void cp_tensorcore(dt *d_X,dt *d_A,dt *d_B,dt *d_C,long a,long b,long c,long r){
// X is a*b*c; A is a*r; B is b*r; C is c*r
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
//	hipblasSetMathMode(handle,HIPBLAS_TENSOR_OP_MATH);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	int *d_info = NULL;
	hipMalloc((void**)&d_info,sizeof(int));
	int *d_Ipiv = NULL; // PA=LU, P is control weather permute
	hipMalloc((void**)&d_Ipiv,sizeof(int));
	int lwork=0;
	dt *d_work = NULL;

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	hiprandGenerateUniform(gen,d_B,b*r);
	hiprandGenerateUniform(gen,d_C,c*r);

	dt *d_X2;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c);
	t2m(d_X,d_X2,2,a,b,c);
	hipDeviceSynchronize();

	half *h_X;
	half *h_X2;
	hipMalloc((void**)&h_X,sizeof(half)*a*b*c);
	hipMalloc((void**)&h_X2,sizeof(half)*a*b*c);
	f2h(d_X,h_X,a*b*c);
	f2h(d_X2,h_X2,a*b*c);
	
	dt *d_ATA,*d_BTB,*d_CTC;
	hipMalloc((void**)&d_ATA,sizeof(dt)*r*r);
	hipMalloc((void**)&d_BTB,sizeof(dt)*r*r);
	hipMalloc((void**)&d_CTC,sizeof(dt)*r*r);

	dt *d_CkrB;
	hipMalloc((void**)&d_CkrB,sizeof(dt)*b*c*r); //GPU store C kr B
	dt *d_CkrA;
	hipMalloc((void**)&d_CkrA,sizeof(dt)*a*c*r); //GPU store C kr A
	dt *d_BkrA;
	hipMalloc((void**)&d_BkrA,sizeof(dt)*a*b*r); //GPU store B kr A

	dt *d_At_r;
	hipMalloc((void**)&d_At_r,sizeof(dt)*a*r); //GPU store (CkrB)'*X1' as right part 
	dt *d_At_l;
	hipMalloc((void**)&d_At_l,sizeof(dt)*r*r); //GPU store (CTC.*BTB)' as left part
	dt *d_Bt_r;
	hipMalloc((void**)&d_Bt_r,sizeof(dt)*b*r); //GPU store (CkrA)'*X2' as right part 
	dt *d_Bt_l;
	hipMalloc((void**)&d_Bt_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	dt *d_Ct_r;
	hipMalloc((void**)&d_Ct_r,sizeof(dt)*c*r); //GPU store (BkrA)'*X3' as right part 
	dt *d_Ct_l;
	hipMalloc((void**)&d_Ct_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	half *h_CkrB,*h_CkrA,*h_BkrA;
	hipMalloc((void**)&h_CkrB,sizeof(half)*c*b*r);
	hipMalloc((void**)&h_CkrA,sizeof(half)*a*c*r);
	hipMalloc((void**)&h_BkrA,sizeof(half)*a*b*r);

	const int L = 500;
for(int i = 0;i<L;i++){
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,c,1,&alpha,d_B,b,b,d_C,c,c,&beta,d_CkrB,b,b*c,r);
	f2h(d_CkrB,h_CkrB,b*c*r);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,a,b*c,&alpha,h_CkrB,HIP_R_16F,b*c,h_X,HIP_R_16F,a,&beta,d_At_r,HIP_R_32F,r,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,c,&alpha,d_C,c,d_C,c,&beta,d_CTC,r);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);
	hadmard(d_CTC,d_BTB,d_At_l,r,r);

	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_At_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_At_l,r,d_work,d_Ipiv,d_info);
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,a,d_At_l,r,d_Ipiv,d_At_r,r,d_info);
	hipDeviceSynchronize();
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,a,r,&alpha,d_At_r,r,&beta,d_A,a,d_A,a);

	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,c,1,&alpha,d_A,a,a,d_C,c,c,&beta,d_CkrA,a,a*c,r);
	f2h(d_CkrA,h_CkrA,a*c*r);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,b,a*c,&alpha,h_CkrA,HIP_R_16F,a*c,h_X2,HIP_R_16F,b,&beta,d_Bt_r,HIP_R_32F,r,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);

	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,a,&alpha,d_A,a,d_A,a,&beta,d_ATA,r);
	hadmard(d_CTC,d_ATA,d_Bt_l,r,r);
	hipDeviceSynchronize();
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Bt_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Bt_l,r,d_work,d_Ipiv,d_info);
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,b,d_Bt_l,r,d_Ipiv,d_Bt_r,r,d_info);
	hipDeviceSynchronize();
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,r,&alpha,d_Bt_r,r,&beta,d_B,b,d_B,b);

	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,a*b,r);
	f2h(d_BkrA,h_BkrA,a*b*r);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,c,b*a,&alpha,h_BkrA,HIP_R_16F,b*a,h_X,HIP_R_16F,a*b,&beta,d_Ct_r,HIP_R_32F,r,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);

	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);
	hadmard(d_BTB,d_ATA,d_Ct_l,r,r);
	hipDeviceSynchronize();
	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Ct_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Ct_l,r,d_work,d_Ipiv,d_info);
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,c,d_Ct_l,r,d_Ipiv,d_Ct_r,r,d_info);
	hipDeviceSynchronize();
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,r,&alpha,d_Ct_r,r,&beta,d_C,c,d_C,c);

	if(i == L-1){
		dt error;
		dt *d_recover;
		hipMalloc((void**)&d_recover,sizeof(dt)*a*b*c);
		gencp(d_recover,d_A,d_B,d_C,a,b,c,r);
		rse(d_X,d_recover,a*b*c,&error);
		cout<<error<<endl;
		hipFree(d_recover);
	}

}
	hipFree(d_At_r);
	hipFree(d_At_l);
	hipFree(d_Bt_r);
	hipFree(d_Bt_l);
	hipFree(d_Ct_r);
	hipFree(d_Ct_l);
	hipDeviceSynchronize();

	hipFree(h_X);hipFree(h_X2);
	hipFree(h_CkrB); hipFree(h_CkrA);hipFree(h_BkrA);
	
	hipFree(d_CkrB);
	hipFree(d_CkrA);
	hipFree(d_BkrA);

	hipFree(d_B);
	hipFree(d_X);
	hipFree(d_C);
	hipFree(d_A);
	hipFree(d_X2);

	hipFree(d_ATA);
	hipFree(d_BTB);
	hipFree(d_CTC);
	hipFree(d_Ipiv);
	hipFree(d_info);
	hipFree(d_work);
	hipsolverDnDestroy(cusolverH);
	hipblasDestroy(handle);
}
