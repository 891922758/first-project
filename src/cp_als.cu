#include "cudecompose.h"

void cp_als(dt *d_X,dt *d_A,dt *d_B,dt *d_C,long a,long b,long c,long r){

// X is a*b*c; A is a*r; B is b*r; C is c*r
// we assume they all store as column
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	int *d_info = NULL;
	hipMalloc((void**)&d_info,sizeof(int));
	int *d_Ipiv = NULL; // PA=LU, P is control weather permute
	hipMalloc((void**)&d_Ipiv,sizeof(int));
	int lwork=0;
	dt *d_work = NULL;

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);
	hiprandGenerateUniform(gen,d_B,b*r);
	hiprandGenerateUniform(gen,d_C,c*r);

	dt *d_X2;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c);
	t2m(d_X,d_X2,2,a,b,c);
	hipDeviceSynchronize();
	
	dt *d_ATA,*d_BTB,*d_CTC;
	hipMalloc((void**)&d_ATA,sizeof(dt)*r*r);
	hipMalloc((void**)&d_BTB,sizeof(dt)*r*r);
	hipMalloc((void**)&d_CTC,sizeof(dt)*r*r);

	dt *d_CkrB;
	hipMalloc((void**)&d_CkrB,sizeof(dt)*b*c*r); //GPU store C kr B
	dt *d_CkrA;
	hipMalloc((void**)&d_CkrA,sizeof(dt)*a*c*r); //GPU store C kr A
	dt *d_BkrA;
	hipMalloc((void**)&d_BkrA,sizeof(dt)*a*b*r); //GPU store B kr A
	const int L = 500;

for(int i = 0;i<L;i++){
	// update A
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,c,1,&alpha,d_B,b,b,d_C,c,c,&beta,d_CkrB,b,b*c,r);
	dt *d_At_r;
	hipMalloc((void**)&d_At_r,sizeof(dt)*a*r); //GPU store (CkrB)'*X1' as right part 
	dt *d_At_l;
	hipMalloc((void**)&d_At_l,sizeof(dt)*r*r); //GPU store (CTC.*BTB)' as left part

	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,a,b*c,&alpha,d_CkrB,b*c,d_X,a,&beta,d_At_r,r);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,c,&alpha,d_C,c,d_C,c,&beta,d_CTC,r);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);

	hadmard(d_CTC,d_BTB,d_At_l,r,r);

	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_At_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_At_l,r,d_work,d_Ipiv,d_info);
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,a,d_At_l,r,d_Ipiv,d_At_r,r,d_info);
	hipDeviceSynchronize();
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,a,r,&alpha,d_At_r,r,&beta,d_A,a,d_A,a);

// update B
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,c,1,&alpha,d_A,a,a,d_C,c,c,&beta,d_CkrA,a,a*c,r);
	dt *d_Bt_r;
	hipMalloc((void**)&d_Bt_r,sizeof(dt)*b*r); //GPU store (CkrA)'*X2' as right part 
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_T,r,b,a*c,&alpha,d_CkrA,a*c,d_X2,b,&beta,d_Bt_r,r);

	dt *d_Bt_l;
	hipMalloc((void**)&d_Bt_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,a,&alpha,d_A,a,d_A,a,&beta,d_ATA,r);
	hadmard(d_CTC,d_ATA,d_Bt_l,r,r);
	hipDeviceSynchronize();

	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Bt_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Bt_l,r,d_work,d_Ipiv,d_info);
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,b,d_Bt_l,r,d_Ipiv,d_Bt_r,r,d_info);
	hipDeviceSynchronize();
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,r,&alpha,d_Bt_r,r,&beta,d_B,b,d_B,b);

//update C
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b,1,&alpha,d_A,a,a,d_B,b,b,&beta,d_BkrA,a,a*b,r);
	dt *d_Ct_r;
	hipMalloc((void**)&d_Ct_r,sizeof(dt)*c*r); //GPU store (BkrA)'*X3' as right part 
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,c,a*b,&alpha,d_BkrA,a*b,d_X,a*b,&beta,d_Ct_r,r);

	dt *d_Ct_l;
	hipMalloc((void**)&d_Ct_l,sizeof(dt)*r*r); //GPU store (CTC.*ATA)' as left part
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r,r,b,&alpha,d_B,b,d_B,b,&beta,d_BTB,r);
	hadmard(d_BTB,d_ATA,d_Ct_l,r,r);
	hipDeviceSynchronize();

	hipsolverDnSgetrf_bufferSize(cusolverH,r,r,d_Ct_l,r,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSgetrf(cusolverH,r,r,d_Ct_l,r,d_work,d_Ipiv,d_info);
	hipsolverDnSgetrs(cusolverH,HIPBLAS_OP_N,r,c,d_Ct_l,r,d_Ipiv,d_Ct_r,r,d_info);
	hipDeviceSynchronize();
	hipblasSgeam(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,r,&alpha,d_Ct_r,r,&beta,d_C,c,d_C,c);

/*	if(i == L-1){
		dt error;
		dt *d_recover;
		hipMalloc((void**)&d_recover,sizeof(dt)*a*b*c);
		gencp(d_recover,d_A,d_B,d_C,a,b,c,r);
		rse(d_X,d_recover,a*b*c,&error);
		cout<<error<<endl;
		hipFree(d_recover);
	}
*/
	hipFree(d_At_r);
	hipFree(d_At_l);
	hipFree(d_Bt_r);
	hipFree(d_Bt_l);
	hipFree(d_Ct_r);
	hipFree(d_Ct_l);
}

	hipDeviceSynchronize();
	hipFree(d_CkrB);
	hipFree(d_CkrA);
	hipFree(d_BkrA);
	hipFree(d_X2);
	hipFree(d_ATA);
	hipFree(d_BTB);
	hipFree(d_CTC);
	hipFree(d_Ipiv);
	hipFree(d_info);
	hipFree(d_work);
	hipsolverDnDestroy(cusolverH);
	hipblasDestroy(handle);
}
