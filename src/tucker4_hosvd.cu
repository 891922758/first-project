#include "cudecompose.h"

void tucker_hosvd4(dt *d_X,dt *d_core,dt *d_U1,dt *d_U2,dt *d_U3,dt *d_U4,long a,long b,long c,long d,long r1,long r2,long r3,long r4){
	//X is a*b*c*d, core is r1*r2*r3*r4, U1 is a*r1,U2 b*r2,U3 is c*r3 U4 is d*r4 
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	dt *d_work = NULL;
	int lwork=0;
	int *devInfo=NULL;
	hipMalloc((void**)&devInfo,sizeof(int));

	dt *d_X1_X1,*d_X2_X2,*d_X3_X3,*d_X4_X4;
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);
	hipMalloc((void**)&d_X4_X4,sizeof(dt)*d*d);
	hipDeviceSynchronize();

	dt *d_X2,*d_Idemat;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c*d);
	hipMalloc((void**)&d_Idemat,sizeof(dt)*a*a);
	initMat(d_Idemat,a);
	hipDeviceSynchronize();
//	cout<<"Idemat"<<endl; printTensor(d_Idemat,a,a,1);
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,b,a,a,&alpha,d_X,a,a*b,d_Idemat,a,0,&beta,d_X2,b,a*b,c*d);
	dt *d_X3,*d_Idemat3;
	hipMalloc((void**)&d_X3,sizeof(dt)*a*b*c*d);
	hipMalloc((void**)&d_Idemat3,sizeof(dt)*a*b*a*b);
	initMat(d_Idemat3,a*b);
	hipDeviceSynchronize();
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,a*b,a*b,&alpha,d_X,a*b,a*b*c,d_Idemat3,a*b,0,&beta,d_X3,c,a*b*c,d);
	hipFree(d_Idemat3);
	hipFree(d_Idemat);

	//compute X1*X1' a*bcd * bcd*a
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b*c*d,&alpha,d_X,a,d_X,a,&beta,d_X1_X1,a);
	//compute X2*X2' b*acd * acd*b
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,b,a*c*d,&alpha,d_X2,b,d_X2,b,&beta,d_X2_X2,b);
	//compute X3*X3' c*abd * abd*c
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,c,c,a*b*d,&alpha,d_X3,c,d_X3,c,&beta,d_X3_X3,c);
	//compute X4*X4' d*abc * abc*d
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,d,d,a*b*c,&alpha,d_X,a*b*c,d_X,a*b*c,&beta,d_X4_X4,d);
	hipDeviceSynchronize();

	// syevd for U1,U2,U3,U4
	dt *d_W1,*d_W2,*d_W3,*d_W4; 
	hipMalloc((void**)&d_W1,sizeof(dt)*a);
	hipMalloc((void**)&d_W2,sizeof(dt)*b);
	hipMalloc((void**)&d_W3,sizeof(dt)*c);
	hipMalloc((void**)&d_W4,sizeof(dt)*d);
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverEigRange_t range = HIPSOLVER_EIG_RANGE_ALL;
	int meig1=a; int meig2=b; int meig3=c; int meig4=d;
	// turn X1X1 to eigvectores and view as U1
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,d_work,lwork,devInfo);
//	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<infogpu<<endl;
	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"first vectores"<<endl; printTensor(d_X1_X1+(a-r1)*a,a,r1,1);

	// turn X2X2 to eigvectores and view as U2
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,d_work,lwork,devInfo);
//	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<infogpu<<endl;
	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"second vectors"<<endl; printTensor(d_X2_X2+(b-r2)*b,b,r2,1);
	
	// turn X3X3 to eigvectores and view as U3
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,d_work,lwork,devInfo);
//	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<infogpu<<endl;
	hipDeviceSynchronize();
	//cout<<"third vectores"<<endl; printTensor(d_X3_X3+(c-r3)*c,c,r3,1);

	// turn X4X4 to eigvectores and view as U4
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,d,d_X4_X4,d,0.0,1e06,1,d,&meig4,d_W4,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,d,d_X4_X4,d,0.0,1e06,1,d,&meig4,d_W4,d_work,lwork,devInfo);
//	hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//	cout<<infogpu<<endl;
	hipDeviceSynchronize();
	//cout<<"third vectores"<<endl; printTensor(d_X4_X4+(d-r4)*d,d,r4,1);
	hipFree(d_W1);
	hipFree(d_W2);
	hipFree(d_W3);
	hipFree(d_W4);

	hipMemcpy(d_U1,d_X1_X1+(a-r1)*a,sizeof(dt)*a*r1,hipMemcpyDeviceToDevice);
	hipMemcpy(d_U2,d_X2_X2+(b-r2)*b,sizeof(dt)*b*r2,hipMemcpyDeviceToDevice);
	hipMemcpy(d_U3,d_X3_X3+(c-r3)*c,sizeof(dt)*c*r3,hipMemcpyDeviceToDevice);
	hipMemcpy(d_U4,d_X4_X4+(d-r4)*d,sizeof(dt)*d*r4,hipMemcpyDeviceToDevice);
	
	// then compute X x1U1 x2U2 x3U3,we need extra two intenal vals and core to store last result 
	// a*b*c  a*r1  b*r2  c*r3
	// X x1U1' =U1'*X1  X1 can obtain direct store as X
	dt *d_XU1,*d_XU1U2,*d_XU1U2U3;
	hipMalloc((void**)&d_XU1,sizeof(dt)*r1*b*c*d);
	hipMalloc((void**)&d_XU1U2,sizeof(dt)*r1*r2*c*d);
	hipMalloc((void**)&d_XU1U2U3,sizeof(dt)*r1*r2*r3*d);
	//X X1 U1 a*b*c*d a*r1
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r1,b*c*d,a,&alpha,d_X1_X1+(a-r1)*a,a,d_X,a,&beta,d_XU1,r1);
	//XU1*U2 r1*b*c*d  b*r2
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,d_XU1,r1,r1*b,d_X2_X2+(b-r2)*b,b,0,&beta,d_XU1U2,r1,r1*r2,c*d);
	//XU1U2U3  r1*r2*c*d c*r3
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2,r3,c,&alpha,d_XU1U2,r1*r2,r1*r2*c,d_X3_X3+(c-r3)*c,c,0,&beta,d_XU1U2U3,r1*r2,r1*r2*r3,d);
	//core  r1*r2*r3*d  d*r4
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2*r3,r4,d,&alpha,d_XU1U2U3,r1*r2*r3,d_X4_X4+(d-r4)*d,d,&beta,d_core,r1*r2*r3);
	hipDeviceSynchronize();

	//recover by X = core X1U1 X2U2 X3U3 X4U4
	dt error= 0.0;
	dt *d_rec;
	hipMalloc((void**)&d_rec,sizeof(dt)*a*b*c*d);
	gentucker4(d_rec,d_core,d_U1,d_U2,d_U3,d_U4,a,b,c,d,r1,r2,r3,r4);
	rse(d_X,d_rec,a*b*c*d,&error);
	cout<<error<<endl;
	hipFree(d_rec);
	hipDeviceSynchronize();

	hipDeviceSynchronize();
	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
	hipFree(d_XU1); hipFree(d_XU1U2); hipFree(d_XU1U2U3);
	hipFree(d_work); hipFree(devInfo);
	hipFree(d_X3); hipFree(d_X2);
	hipFree(d_X1_X1); hipFree(d_X2_X2); hipFree(d_X3_X3); hipFree(d_X4_X4);

}


