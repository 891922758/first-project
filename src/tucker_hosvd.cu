#include "cudecompose.h"

void tucker_hosvd(dt *d_X,dt *d_core,dt *d_U1,dt *d_U2,dt *d_U3,long a,long b,long c,long r1,long r2,long r3){
	//X is a*b*c, core is r1*r2*r3, U1 is a*r1,U2 b*r2,U3 is c*r3 
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	dt *d_work = NULL;
	int lwork=0;
	int *devInfo=NULL;
	hipMalloc((void**)&devInfo,sizeof(int));

	dt *d_X1_X1,*d_X2_X2,*d_X3_X3;
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);
	hipDeviceSynchronize();

	dt *d_X2;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c);
	t2m(d_X,d_X2,2,a,b,c);
	hipDeviceSynchronize();

	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b*c,&alpha,d_X,a,d_X,a,&beta,d_X1_X1,a);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,b,a*c,&alpha,d_X2,b,d_X2,b,&beta,d_X2_X2,b);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,c,a*b,&alpha,d_X,a*b,d_X,a*b,&beta,d_X3_X3,c);
	hipDeviceSynchronize();
	
	// syevd for U1,U2,U3
	//data prepare for store eigvalue and eigvectors,we only fetch r1 r2 and r3 eigvectors from origin
	dt *d_W1,*d_W2,*d_W3; 
	hipMalloc((void**)&d_W1,sizeof(dt)*a);
	hipMalloc((void**)&d_W2,sizeof(dt)*b);
	hipMalloc((void**)&d_W3,sizeof(dt)*c);
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverEigRange_t range = HIPSOLVER_EIG_RANGE_ALL;
	int meig1=a; int meig2=b; int meig3=c;
	// turn X1X1 to eigvectores and view as U1
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,d_work,lwork,devInfo);
	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"first vectores"<<endl; printTensor(d_X1_X1+(a-r1)*a,a,r1,1);

	// turn X2X2 to eigvectores and view as U2
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,d_work,lwork,devInfo);
	hipFree(d_work);
	hipDeviceSynchronize();
	//cout<<"second vectors"<<endl; printTensor(d_X2_X2+(b-r2)*b,b,r2,1);
	
	// turn X3X3 to eigvectores and view as U3
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,d_work,lwork,devInfo);
	hipFree(d_work);
	//cout<<"third vectores"<<endl; printTensor(d_X3_X3+(c-r3)*c,c,r3,1);
	hipFree(d_W1);
	hipFree(d_W2);
	hipFree(d_W3);

	hipMemcpy(d_U1,d_X1_X1+(a-r1)*a,sizeof(dt)*a*r1,hipMemcpyDeviceToDevice);
	hipMemcpy(d_U2,d_X2_X2+(b-r2)*b,sizeof(dt)*b*r2,hipMemcpyDeviceToDevice);
	hipMemcpy(d_U3,d_X3_X3+(c-r3)*c,sizeof(dt)*c*r3,hipMemcpyDeviceToDevice);

	// then compute X x1U1 x2U2 x3U3,we need extra two intenal vals and core to store last result 
	// a*b*c  a*r1  b*r2  c*r3
	// X x1U1' =U1'*X1  X1 can obtain direct store as X
	dt *d_XU1,*d_XU1U2;
	hipMalloc((void**)&d_XU1,sizeof(dt)*r1*b*c);
	hipMalloc((void**)&d_XU1U2,sizeof(dt)*r1*r2*c);
	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r1,b*c,a,&alpha,d_U1,a,d_X,a,&beta,d_XU1,r1);
	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,d_XU1,r1,r1*b,d_U2,b,0,&beta,d_XU1U2,r1,r1*r2,c);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2,r3,c,&alpha,d_XU1U2,r1*r2,d_U3,c,&beta,d_core,r1*r2);
	hipDeviceSynchronize();
/*
	//recover by X = core X1U1 X2U2 X3U3
	// r1*r2*r3  a*r1 ,b*r2 ,c*r3
	dt error= 0.0;
	dt *d_rec;
	hipMalloc((void**)&d_rec,sizeof(dt)*a*b*c);
	gentucker(d_rec,d_core,d_U1,d_U2,d_U3,a,b,c,r1,r2,r3);
	rse(d_X,d_rec,a*b*c,&error);
	cout<<error<<endl;
	hipFree(d_rec);
	hipDeviceSynchronize();
*/
	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
	hipFree(d_XU1);
	hipFree(d_XU1U2);
	hipFree(devInfo);
	hipFree(d_X1_X1);
	hipFree(d_X2_X2);
	hipFree(d_X3_X3);
	hipFree(d_X2);
}

