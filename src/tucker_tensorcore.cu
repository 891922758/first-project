#include "cudecompose.h"

void tucker_tensorcore(dt *d_X,dt *d_core,dt *d_U1,dt *d_U2,dt *d_U3,long a,long b,long c,long r1,long r2,long r3){
	//X is a*b*c, core is r1*r2*r3, U1 is a*r1,U2 b*r2,U3 is c*r3 
	dt alpha = 1.0;
	dt beta = 0.0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
	dt *d_work = NULL;
	int lwork=0;
	int *devInfo=NULL;
	hipMalloc((void**)&devInfo,sizeof(int));

	dt *d_X1_X1,*d_X2_X2,*d_X3_X3;
	hipMalloc((void**)&d_X1_X1,sizeof(dt)*a*a);
	hipMalloc((void**)&d_X2_X2,sizeof(dt)*b*b);
	hipMalloc((void**)&d_X3_X3,sizeof(dt)*c*c);
	hipDeviceSynchronize();

	dt *d_X2;
	hipMalloc((void**)&d_X2,sizeof(dt)*a*b*c);
	t2m(d_X,d_X2,2,a,b,c);
	hipDeviceSynchronize();
	half *h_X,*h_X2;
	hipMalloc((void**)&h_X2,sizeof(half)*a*b*c);
	hipMalloc((void**)&h_X,sizeof(half)*a*b*c);
	f2h(d_X,h_X,a*b*c);
	f2h(d_X2,h_X2,a*b*c);
	hipFree(d_X2);
	hipDeviceSynchronize();

	//compute X1*X1' a*bc * bc*a
	hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b*c,&alpha,h_X,HIP_R_16F,a,h_X,HIP_R_16F,a,&beta,d_X1_X1,HIP_R_32F,a,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,a,b*c,&alpha,d_X,a,d_X,a,&beta,d_X1_X1,a);
//	cout<<"X1*X1'"<<endl; printTensor(d_X1_X1,2,3,1);
	//compute X2*X2' b*ac * ac*b
//	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,b,a*c,&alpha,d_X2,b,d_X2,b,&beta,d_X2_X2,b);
	hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,b,b,a*c,&alpha,h_X2,HIP_R_16F,b,h_X2,HIP_R_16F,b,&beta,d_X2_X2,HIP_R_32F,b,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	cout<<"X2*X2'"<<endl; printTensor(d_X2_X2,2,3,1);
	//compute X3*X3' c*ab * ab*c
//	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,c,a*b,&alpha,d_X,a*b,d_X,a*b,&beta,d_X3_X3,c);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,c,c,b*a,&alpha,h_X,HIP_R_16F,b*a,h_X,HIP_R_16F,a*b,&beta,d_X3_X3,HIP_R_32F,c,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	cout<<"X3*X3'"<<endl; printTensor(d_X3_X3,2,3,1);
	hipDeviceSynchronize();
	
	// syevd for U1,U2,U3
	//data prepare for store eigvalue and eigvectors,we only fetch r1 r2 and r3 eigvectors from origin
	dt *d_W1,*d_W2,*d_W3; 
	hipMalloc((void**)&d_W1,sizeof(dt)*a);
	hipMalloc((void**)&d_W2,sizeof(dt)*b);
	hipMalloc((void**)&d_W3,sizeof(dt)*c);
	hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
	hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
	hipsolverEigRange_t range = HIPSOLVER_EIG_RANGE_ALL;
	int meig1=a; int meig2=b; int meig3=c;
	// turn X1X1 to eigvectores and view as U1
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,a,d_X1_X1,a,0.0,1e06,1,a,&meig1,d_W1,d_work,lwork,devInfo);
//hipMemcpy(&infogpu,devInfo,sizeof(int),hipMemcpyDeviceToHost);
//cout<<infogpu<<endl;
	hipFree(d_work);
	hipDeviceSynchronize();
//cout<<"first vectores"<<endl; printTensor(d_X1_X1+(a-r1)*a,2,3,1);

	// turn X2X2 to eigvectores and view as U2
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,b,d_X2_X2,b,0.0,1e06,1,b,&meig2,d_W2,d_work,lwork,devInfo);
	hipFree(d_work);
	hipDeviceSynchronize();
	
	// turn X3X3 to eigvectores and view as U3
	hipsolverDnSsyevdx_bufferSize(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,&lwork);
	hipMalloc((void**)&d_work,sizeof(dt)*lwork);
	hipsolverDnSsyevdx(cusolverH,jobz,range,uplo,c,d_X3_X3,c,0.0,1e06,1,c,&meig3,d_W3,d_work,lwork,devInfo);
	hipFree(d_work);
	hipFree(d_W1);
	hipFree(d_W2);
	hipFree(d_W3);
	
	half *h_U1,*h_U2,*h_U3;
	hipMalloc((void**)&h_U1,sizeof(half)*a*r1);
	hipMalloc((void**)&h_U2,sizeof(half)*b*r2);
	hipMalloc((void**)&h_U3,sizeof(half)*c*r3);
	f2h(d_X1_X1+(a-r1)*a,h_U1,a*r1);
	f2h(d_X2_X2+(b-r2)*b,h_U2,b*r2);
	f2h(d_X3_X3+(c-r3)*c,h_U3,c*r3);
	hipDeviceSynchronize();
	// then compute X x1U1 x2U2 x3U3,we need extra two intenal vals and core to store last result 
	// a*b*c  a*r1  b*r2  c*r3
	// X x1U1' =U1'*X1  X1 can obtain direct store as X
	half *h_XU1,*h_XU1U2;
	hipMalloc((void**)&h_XU1,sizeof(half)*r1*b*c);
	hipMalloc((void**)&h_XU1U2,sizeof(half)*r1*r2*c);
	//X X1 U1 a*b*c a*r1
//	hipblasSgemm(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r1,b*c,a,&alpha,d_X1_X1+(a-r1)*a,a,d_X,a,&beta,d_XU1,r1);
	hipblasGemmEx(handle,HIPBLAS_OP_T,HIPBLAS_OP_N,r1,b*c,a,&alpha,h_U1,HIP_R_16F,a,h_X,HIP_R_16F,a,&beta,h_XU1,HIP_R_16F,r1,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	cout<<"XU1"<<endl; printTensor(d_XU1,2,3,1);
	//XU1*U2 r1*b *c  b*r2
//	hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,d_XU1,r1,r1*b,d_X2_X2+(b-r2)*b,b,0,&beta,d_XU1U2,r1,r1*r2,c);
	hipblasGemmStridedBatchedEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1,r2,b,&alpha,h_XU1,HIP_R_16F,r1,r1*b,h_U2,HIP_R_16F,b,0,&beta,h_XU1U2,HIP_R_16F,r1,r1*r2,c,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
	//cout<<"XU1U2"<<endl; printTensor(d_XU1U2,r1,r2*c,1);
	//XU1U2*U3'  r1*r2*r3 c*r3
//	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2,r3,c,&alpha,d_XU1U2,r1*r2,d_X3_X3+(c-r3)*c,c,&beta,d_core,r1*r2);
	hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,r1*r2,r3,c,&alpha,h_XU1U2,HIP_R_16F,r1*r2,h_U3,HIP_R_16F,c,&beta,d_core,HIP_R_32F,r1*r2,HIP_R_32F,CUBLAS_GEMM_DEFAULT_TENSOR_OP);
//	cout<<"core"<<endl; printTensor(d_core,1,r1*r2*r3,1);
//	cout<<"core"<<endl; printTensor(d_core,1,5,1);
	hipDeviceSynchronize();


	//recover by X = core X1U1 X2U2 X3U3
	// r1*r2*r3  a*r1 ,b*r2 ,c*r3
	dt error= 0.0;
	dt *d_rec;
	hipMalloc((void**)&d_rec,sizeof(dt)*a*b*c);
	gentucker(d_rec,d_core,d_X1_X1+(a-r1)*a,d_X2_X2+(b-r2)*b,d_X3_X3+(c-r3)*c,a,b,c,r1,r2,r3);
	rse(d_X,d_rec,a*b*c,&error);
	cout<<error<<endl;
	hipFree(d_rec);
	hipDeviceSynchronize();

	hipblasDestroy(handle);
	hipsolverDnDestroy(cusolverH);
	hipFree(h_U1);
	hipFree(h_U2);
	hipFree(h_U3);

	hipFree(h_XU1);
	hipFree(h_XU1U2);
	hipFree(devInfo);
	hipFree(d_X1_X1);
	hipFree(d_X2_X2);
	hipFree(d_X3_X3);
	hipFree(h_X);
	hipFree(h_X2);
}

